#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BeamChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "TNNets.h"
#include "BeamChunker.h"
#include "BatchBeamDecoder.h"
#include "Evalb.h"

#include "Example.h"

BeamChunker::BeamChunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
}

BeamChunker::BeamChunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
}

BeamChunker::~BeamChunker() {
}
    
std::pair<BeamChunker::ChunkedResultType, BeamChunker::ChunkedResultType> BeamChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<XPU> &modelParas) {
    const int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transSystemPtr->getActNumber();
    const int beam_size = m_nBeamSize;

    static int longestLen = (*std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2){ return inst1.input.size() < inst2.input.size(); })).input.size();
    static const int nMaxLatticeSize = (m_nBeamSize + 1) * longestLen;
    int threads_num = CConfig::nThread;

    std::vector<State *> lattices(threads_num);
    std::vector<State **> lattice_indexes(threads_num);
    for (int i = 0; i < threads_num; i++) {
        lattices[i] = new State[nMaxLatticeSize];
        lattice_indexes[i] = new State*[longestLen + 2];
    }
    // std::vector<Model<XPU>> models;
    // for (int i = 0; i < threads_num; i++) {
    //     models.push_back(modelParas);
    // }

    static int chunkRound = 1;

    clock_t start, end;
    start = clock();
    ChunkedDataSet predictDevSet(goldDevSet.size());

    std::vector<ChunkedDataSet> threadPredictDevSets(threads_num);

#pragma omp parallel num_threads(threads_num)
    {
        int threadIndex = omp_get_thread_num();

        // TNNets tnnets(beam_size, num_in, num_hidden, num_out, &models[threadIndex], false);
        TNNets tnnets(beam_size, num_in, num_hidden, num_out, &modelParas, false);
        ChunkedDataSet &threadPredictDevSet = threadPredictDevSets[threadIndex];

        for (unsigned inst = threadIndex; inst < static_cast<unsigned>(devInstances.size()); inst += threads_num) {
            LabeledSequence predictSent(devInstances[inst].input);

            BeamDecoder decoder(&(devInstances[inst]), 
                                m_transSystemPtr,
                                m_featManagerPtr,
                                m_featEmbManagerPtr,
                                m_nBeamSize, 
                                lattices[threadIndex],
                                lattice_indexes[threadIndex],
                                false);

            decoder.generateLabeledSequence(tnnets, predictSent);

            threadPredictDevSet.push_back(predictSent);
        }
    }
    for (int i = 0; i < threads_num; i++) {
        delete []lattices[i];
        delete []lattice_indexes[i];
    }

    for (int i = 0; i < threads_num; i++) {
        for (int j = 0; j < threadPredictDevSets[i].size(); j++) {
            predictDevSet[i + j * threads_num] = threadPredictDevSets[i][j];
        }
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, \ttime: " << time_used << " \taverage: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    auto FB1 = Evalb::eval(predictDevSet, goldDevSet);
    auto NPFB1 = Evalb::eval(predictDevSet, goldDevSet, true);

    return std::make_pair(FB1, NPFB1);
}

void BeamChunker::generateMultiThreadsMiniBatchData(std::vector<std::vector<GlobalExample *>> &multiThread_miniBatch_data) {
    std::random_shuffle(gExamples.begin(), gExamples.end());

    // prepare mini-batch data for each threads
    static int exampleNumOfThread = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()))/ CConfig::nThread;

    auto sp = gExamples.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        std::vector<GlobalExample *> threadExamples;
        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(&(*p));
        }
        sp = ep;
        ep += exampleNumOfThread;
        multiThread_miniBatch_data.push_back(threadExamples);
    }
}

void BeamChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "[trainingSet involved initing]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[devSet involved initing]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    const int batch_size = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()));

    InitTensorEngine<XPU>();
    Stream<XPU> *sstream = m_modelPtr->stream;

    auto featureTypes = m_featManagerPtr->getFeatureTypes();

    Model<XPU> &modelParas = *(m_modelPtr.get());
    Model<XPU> adaGradSquares(num_in, num_hidden, num_out, featureTypes, sstream);

    auto longestSentence = *std::max_element(gExamples.begin(), gExamples.end(), [](GlobalExample &ge1, GlobalExample &ge2) { return ge1.instance.input.size() < ge2.instance.input.size();} );
    const int longestLen = longestSentence.instance.input.size();

    BatchBeamDecoderMemoryManager decoderMemoryManager(m_nBeamSize, CConfig::nBeamBatchDecoderItemSize, longestLen, CConfig::nThread);
    TNNetsMemoryManager nnetsMemoryManager(CConfig::nThread, longestLen, m_nBeamSize * CConfig::nBeamBatchDecoderItemSize, num_in, num_hidden, num_out, &modelParas);

    ChunkedResultType bestDevFB1 = std::make_tuple(0.0, 0.0, -1.0);
    ChunkedResultType bestDevNPFB1 = std::make_tuple(0.0, 0.0, -1.0);
    clock_t start, end;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (CConfig::saveModel && iter % CConfig::nSaveModelPerIters == 0) {
            saveChunker(iter);
        }
        if (iter % CConfig::nEvaluatePerIters == 0) {
            auto res = chunk(devSet, devGoldSet, modelParas);
            ChunkedResultType &currentFB1 = std::get<0>(res);
            ChunkedResultType &currentNPFB1 = std::get<1>(res);

            if (std::get<2>(currentFB1) > std::get<2>(bestDevFB1)) {
                bestDevFB1 = currentFB1;
                bestDevNPFB1 = currentNPFB1;
                saveChunker(0);
            }
            // if (std::get<2>(currentNPFB1) > std::get<2>(bestDevNPFB1)) {
            //     bestDevNPFB1 = currentNPFB1;
            // }
            auto sf = std::cerr.flags();
            auto sp = std::cerr.precision();
            std::cerr.flags(std::ios::fixed);
            std::cerr.precision(2);
            std::cerr << "current iteration FB1-score  : " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentFB1) << "/" << std::get<1>(currentFB1) << "/" << std::get<2>(currentFB1) << "\t best FB1-score  : " << std::get<0>(bestDevFB1) << "/" << std::get<1>(bestDevFB1) << "/" << std::get<2>(bestDevFB1) << std::endl;
            std::cerr << "current iteration NPFB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentNPFB1) << "/" << std::get<1>(currentNPFB1) << "/" << std::get<2>(currentNPFB1) << "\t best NPFB1-score: " << std::get<0>(bestDevNPFB1)  << "/" << std::get<1>(bestDevNPFB1) << "/" << std::get<2>(bestDevNPFB1) << std::endl;
            std::cerr.flags(sf);
            std::cerr.precision(sp);
        }

        start = clock();

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<XPU> batchCumulatedGrads(num_in, num_hidden, num_out, featureTypes, sstream);
        // begin to multi thread Training
#pragma omp parallel num_threads(CConfig::nThread)
        {
            int threadIndex = omp_get_thread_num();
            // int threadIndex = 0;
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            Model<XPU> cumulatedGrads(num_in, num_hidden, num_out, featureTypes, sstream);

            for (int insti = 0; insti < currentThreadData.size(); insti += CConfig::nBeamBatchDecoderItemSize) {
                std::vector<GlobalExample *> gExamplePtrs;
                std::vector<Instance *> instPtrs;
                for (int i = 0; i < CConfig::nBeamBatchDecoderItemSize; i++) {
                    gExamplePtrs.push_back(currentThreadData[insti + i]);
                    instPtrs.push_back(&(currentThreadData[insti + i]->instance));
                }

                TNNets tnnets(m_nBeamSize * CConfig::nBeamBatchDecoderItemSize, num_in, num_hidden, num_out, &modelParas, nnetsMemoryManager.getNetPtrVec(threadIndex));

                BatchBeamDecoder decoder(
                        instPtrs, 
                        m_transSystemPtr,
                        m_featManagerPtr,
                        m_featEmbManagerPtr,
                        m_nBeamSize,
                        decoderMemoryManager.getLatticePtrVec(threadIndex),
                        decoderMemoryManager.getLatticeIndexPtrVec(threadIndex),
                        true
                        );

                std::vector<State *> predStates = decoder.decode(tnnets, gExamplePtrs);

                tnnets.updateTNNetParas(&cumulatedGrads, decoder);
            } // end for instance traverse

#pragma omp barrier
#pragma omp critical
            batchCumulatedGrads.mergeModel(&cumulatedGrads);

        } // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        end = clock();
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double time_used = (double)(end - start) / CLOCKS_PER_SEC;

            std::cerr << "[" << iter << "] totally train " << batch_size << " sentences, \ttime: " << time_used << " \taverage: " << batch_size / time_used << " sentences/second!" << std::endl; 
        }
    } // end total iteration

    DeleteStream(sstream);
    ShutdownTensorEngine<XPU>();
}

void BeamChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void BeamChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_featManagerPtr.reset(new FeatureManager());
    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager());
    m_transSystemPtr.reset(new ActionStandardSystem());
    if (CConfig::loadModel){
        std::ifstream dict_is(CConfig::strModelDirPath + "/dictionarymanager.model");
        m_dictManagerPtr->loadDictManager(dict_is);

        std::ifstream featManager_is(CConfig::strModelDirPath + "/featuremanager.model");
        m_featManagerPtr->loadFeatureManager(featManager_is, m_dictManagerPtr);

        std::ifstream trans_is(CConfig::strModelDirPath + "/actionsystem.model");
        m_transSystemPtr->loadActionSystem(trans_is);
    } else {
        m_dictManagerPtr->init(goldSet);
        m_featManagerPtr->init(goldSet, m_dictManagerPtr);
        m_transSystemPtr->init(goldSet);
    }

    m_featEmbManagerPtr->init(m_featManagerPtr);

    num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    num_hidden = CConfig::nHiddenSize;
    num_out = m_transSystemPtr->getActNumber();

    srand(0);

    Stream<XPU> *sstream = NewStream<XPU>();

    m_modelPtr.reset(new Model<XPU>(num_in, num_hidden, num_out, m_featEmbManagerPtr->getFeatureTypes(), sstream));
    if (CConfig::loadModel) {
        std::ifstream model_is(CConfig::strModelDirPath + "/netmodel.model");
        m_modelPtr->loadModel(model_is);
    } else {
        m_modelPtr->randomInitialize();
    }

    if (!CConfig::loadModel && CConfig::bReadPretrain) {
        m_featEmbManagerPtr->readPretrainedEmbeddings(*(m_modelPtr.get()));
    }


    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    auto featureTypes = m_featManagerPtr->getFeatureTypes();

    std::cerr << "  total input embedding dim: " << m_featEmbManagerPtr->getTotalFeatEmbSize() << std::endl;
    std::cerr << std::endl << "  train set size: " << trainSet.size() << std::endl;
    std::cerr << "  [begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "    " << ft.typeName << ":" << std::endl;
        std::cerr << "      dictSize = " << ft.dictSize << std::endl;
        std::cerr << "      featSize = " << ft.featSize << std::endl;
        std::cerr << "      embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "  [end]" << std::endl;
}

void BeamChunker::saveChunker(int round) {
    std::string dir = CConfig::strModelDirPath;
    std::string app_str;

    if (round != -1) {
        app_str = "." + std::to_string(round);
    }

    std::ofstream actionSystemOs(dir + "/actionsystem.model" + app_str);
    m_transSystemPtr->saveActionSystem(actionSystemOs);

    std::ofstream dictOs(dir + "/dictionarymanager.model" + app_str);
    m_dictManagerPtr->saveDictManager(dictOs);

    std::ofstream featManagerOs(dir + "/featuremanager.model" + app_str);
    m_featManagerPtr->saveFeatureManager(featManagerOs);

    std::ofstream modelOs(dir + "/netmodel.model" + app_str);
    m_modelPtr->saveModel(modelOs);
}
