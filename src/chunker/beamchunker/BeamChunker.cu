#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BeamChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "TNNets.h"
#include "BeamChunker.h"
#include "Evalb.h"

#include "Example.h"

BeamChunker::BeamChunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
}

BeamChunker::BeamChunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
}

BeamChunker::~BeamChunker() {
}
    
std::pair<double, double> BeamChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<XPU> &modelParas) {
    const int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transSystemPtr->getActNumber();
    const int beam_size = CConfig::nBeamSize;
    static int chunkRound = 1;

    TNNets tnnets(beam_size, num_in, num_hidden, num_out, &modelParas, false);

    clock_t start, end;
    start = clock();
    ChunkedDataSet predictDevSet;
    for (unsigned inst = 0; inst < static_cast<unsigned>(devInstances.size()); inst++) {
        LabeledSequence predictSent(devInstances[inst].input);

        BeamDecoder decoder(&(devInstances[inst]), 
                            m_transSystemPtr,
                            m_featManagerPtr,
                            m_featEmbManagerPtr,
                            m_nBeamSize, 
                            false);

        decoder.generateLabeledSequence(tnnets, predictSent);

        predictDevSet.push_back(predictSent);
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    auto res = Evalb::eval(predictDevSet, goldDevSet);
    double FB1 = std::get<2>(res);
    res = Evalb::eval(predictDevSet, goldDevSet);
    double NPFB1 = std::get<2>(res);

    return std::make_pair(FB1, NPFB1);
}

void BeamChunker::generateMultiThreadsMiniBatchData(std::vector<std::vector<GlobalExample *>> &multiThread_miniBatch_data) {
    std::random_shuffle(gExamples.begin(), gExamples.end());

    // prepare mini-batch data for each threads
    static int exampleNumOfThread = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()))/ CConfig::nThread;

    auto sp = gExamples.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        std::vector<GlobalExample *> threadExamples;
        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(&(*p));
        }
        sp = ep;
        ep += exampleNumOfThread;
        multiThread_miniBatch_data.push_back(threadExamples);
    }
}

void BeamChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "[trainingSet involved initing]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[devSet involved initing]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    const int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transSystemPtr->getActNumber();
    const int batch_size = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()));

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    InitTensorEngine<XPU>();
    std::vector<Stream<XPU> *> streams(CConfig::nThread + 1);

    auto featureTypes = m_featManagerPtr->getFeatureTypes();
    std::cerr << "[begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "  " << ft.typeName << ":" << std::endl;
        std::cerr << "    dictSize = " << ft.dictSize << std::endl;
        std::cerr << "    featSize = " << ft.featSize << std::endl;
        std::cerr << "    embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "[end]" << std::endl;

    Model<XPU> modelParas(num_in, num_hidden, num_out, featureTypes, streams[0], true);
    m_featEmbManagerPtr->readPretrainedEmbeddings(modelParas);
    Model<XPU> adaGradSquares(num_in, num_hidden, num_out, featureTypes, streams[0], false);

    double bestDevFB1 = -1.0;
    double bestDevNPFB1 = -1.0;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            auto res = chunk(devSet, devGoldSet, modelParas);
            double currentFB1 = std::get<0>(res);
            double currentNPFB1 = std::get<1>(res);

            if (currentFB1 > bestDevFB1) {
                bestDevFB1 = currentFB1;
            }
            if (currentNPFB1 > bestDevNPFB1) {
                bestDevNPFB1 = currentNPFB1;
            }
            auto sf = std::cerr.flags();
            auto sp = std::cerr.precision();
            std::cerr.flags(std::ios::fixed);
            std::cerr.precision(2);
            std::cerr << "current iteration FB1-score  : " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentFB1 << "\t   best FB1-score: " << bestDevFB1 << std::endl;
            std::cerr << "current iteration NPFB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentNPFB1 << "\t best NPFB1-score: " << bestDevNPFB1 << std::endl;
            std::cerr.flags(sf);
            std::cerr.precision(sp);
        }

        clock_t start, end;
        start = clock();

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<XPU> batchCumulatedGrads(num_in, num_hidden, num_out, featureTypes, streams[0], false);
        // begin to multi thread Training
#pragma omp parallel
        {
            int threadIndex = omp_get_thread_num();
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            Model<XPU> cumulatedGrads(num_in, num_hidden, num_out, featureTypes, streams[threadIndex + 1], false);

            // for evary instance in this mini-batch
            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                // fetch a to-be-trained instance
                GlobalExample *example = currentThreadData[inst];

                TNNets tnnets(m_nBeamSize, num_in, num_hidden, num_out, &modelParas);

                // decode and update
                // std::cerr << "begin to decode!" << std::endl;
                BeamDecoder decoder(&(example->instance), 
                                    m_transSystemPtr,
                                    m_featManagerPtr,
                                    m_featEmbManagerPtr,
                                    m_nBeamSize, 
                                    true);

                State * predState = decoder.decode(tnnets, example);

                tnnets.updateTNNetParas(&cumulatedGrads, decoder.beam, decoder.bEarlyUpdate, decoder.nGoldTransitionIndex, decoder.goldScoredTran);
            } // end for instance traverse

#pragma omp barrier
#pragma omp critical
            batchCumulatedGrads.mergeModel(&cumulatedGrads);
        } // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        end = clock();
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double time_used = (double)(end - start) / CLOCKS_PER_SEC;

            std::cerr << "[" << iter << "] totally train " << batch_size << " sentences, time: " << time_used << " average: " << batch_size / time_used << " sentences/second!" << std::endl; 
        }
    } // end total iteration

    ShutdownTensorEngine<XPU>();
}

void BeamChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void BeamChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_dictManagerPtr->init(goldSet);

    m_featManagerPtr.reset(new FeatureManager());
    m_featManagerPtr->init(goldSet, m_dictManagerPtr);

    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager(
                m_featManagerPtr->getFeatureTypes(),
                m_featManagerPtr->getDictManagerPtrs(),
                static_cast<real_t>(CConfig::fInitRange)
                ));

    m_transSystemPtr.reset(new ActionStandardSystem());
    m_transSystemPtr->init(goldSet);

    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    std::cerr << std::endl << "  train set size: " << trainSet.size() << std::endl;
}
