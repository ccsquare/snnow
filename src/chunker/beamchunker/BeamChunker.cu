#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BeamChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "TNNets.h"
#include "BeamChunker.h"
#include "Evalb.h"

#include "Example.h"

BeamChunker::BeamChunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
}

BeamChunker::BeamChunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
}

BeamChunker::~BeamChunker() {
}
    
std::pair<BeamChunker::ChunkedResultType, BeamChunker::ChunkedResultType> BeamChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<XPU> &modelParas) {
    const int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transSystemPtr->getActNumber();
    const int beam_size = m_nBeamSize;

    static int longestLen = (*std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2){ return inst1.input.size() < inst2.input.size(); })).input.size();
    static const int nMaxLatticeSize = (m_nBeamSize + 1) * longestLen;
    int threads_num = CConfig::nThread;

    std::vector<State *> lattices(threads_num);
    std::vector<State **> lattice_indexes(threads_num);
    for (int i = 0; i < threads_num; i++) {
        lattices[i] = new State[nMaxLatticeSize];
        lattice_indexes[i] = new State*[longestLen + 2];
    }
    // std::vector<Model<XPU>> models;
    // for (int i = 0; i < threads_num; i++) {
    //     models.push_back(modelParas);
    // }

    static int chunkRound = 1;

    clock_t start, end;
    start = clock();
    ChunkedDataSet predictDevSet(goldDevSet.size());

    std::vector<ChunkedDataSet> threadPredictDevSets(threads_num);

#pragma omp parallel num_threads(threads_num)
    {
        int threadIndex = omp_get_thread_num();

        // TNNets tnnets(beam_size, num_in, num_hidden, num_out, &models[threadIndex], false);
        TNNets tnnets(beam_size, num_in, num_hidden, num_out, &modelParas, false);
        ChunkedDataSet &threadPredictDevSet = threadPredictDevSets[threadIndex];

        for (unsigned inst = threadIndex; inst < static_cast<unsigned>(devInstances.size()); inst += threads_num) {
            LabeledSequence predictSent(devInstances[inst].input);

            BeamDecoder decoder(&(devInstances[inst]), 
                                m_transSystemPtr,
                                m_featManagerPtr,
                                m_featEmbManagerPtr,
                                m_nBeamSize, 
                                lattices[threadIndex],
                                lattice_indexes[threadIndex],
                                false);

            decoder.generateLabeledSequence(tnnets, predictSent);

            threadPredictDevSet.push_back(predictSent);
        }
    }
    for (int i = 0; i < threads_num; i++) {
        delete []lattices[i];
        delete []lattice_indexes[i];
    }

    for (int i = 0; i < threads_num; i++) {
        for (int j = 0; j < threadPredictDevSets[i].size(); j++) {
            predictDevSet[i + j * threads_num] = threadPredictDevSets[i][j];
        }
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, \ttime: " << time_used << " \taverage: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    auto FB1 = Evalb::eval(predictDevSet, goldDevSet);
    auto NPFB1 = Evalb::eval(predictDevSet, goldDevSet, true);

    return std::make_pair(FB1, NPFB1);
}

void BeamChunker::generateMultiThreadsMiniBatchData(std::vector<std::vector<GlobalExample *>> &multiThread_miniBatch_data) {
    std::random_shuffle(gExamples.begin(), gExamples.end());

    // prepare mini-batch data for each threads
    static int exampleNumOfThread = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()))/ CConfig::nThread;

    auto sp = gExamples.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        std::vector<GlobalExample *> threadExamples;
        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(&(*p));
        }
        sp = ep;
        ep += exampleNumOfThread;
        multiThread_miniBatch_data.push_back(threadExamples);
    }
}

void BeamChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "[trainingSet involved initing]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[devSet involved initing]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    const int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transSystemPtr->getActNumber();
    const int batch_size = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()));

    srand(0);

    InitTensorEngine<XPU>();
    Stream<XPU> *sstream = NewStream<XPU>();

    auto featureTypes = m_featManagerPtr->getFeatureTypes();
    std::cerr << "[begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "  " << ft.typeName << ":" << std::endl;
        std::cerr << "    dictSize = " << ft.dictSize << std::endl;
        std::cerr << "    featSize = " << ft.featSize << std::endl;
        std::cerr << "    embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "[end]" << std::endl;

    Model<XPU> modelParas(num_in, num_hidden, num_out, m_featEmbManagerPtr, sstream, true);
    Model<XPU> adaGradSquares(num_in, num_hidden, num_out, m_featEmbManagerPtr, sstream, false);

    auto longestSentence = *std::max_element(gExamples.begin(), gExamples.end(), [](GlobalExample &ge1, GlobalExample &ge2) { return ge1.instance.input.size() < ge2.instance.input.size();} );
    const int longestLen = longestSentence.instance.input.size();
    const int nMaxLatticeSize = (m_nBeamSize + 1) * longestLen;
    std::vector<State *> lattices(CConfig::nThread);
    std::vector<State **> lattice_indexes(CConfig::nThread);
    for (int i = 0; i < CConfig::nThread; i++) {
        lattices[i] = new State[nMaxLatticeSize];
        lattice_indexes[i] = new State*[longestLen + 2];
    }
    std::vector<std::vector<NNet<XPU> *>> netss;
    for (int i = 0; i < CConfig::nThread; i++) {
        std::vector<NNet<XPU> *> nets(longestLen + 1);

        for (int j = 0; j < nets.size(); j++) {
            nets[j] = new NNet<XPU>(m_nBeamSize, num_in, num_hidden, num_out, &modelParas);
        }

        netss.push_back(nets);
    }

    ChunkedResultType bestDevFB1 = std::make_tuple(0.0, 0.0, -1.0);
    ChunkedResultType bestDevNPFB1 = std::make_tuple(0.0, 0.0, -1.0);
    clock_t start, end;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            auto res = chunk(devSet, devGoldSet, modelParas);
            ChunkedResultType &currentFB1 = std::get<0>(res);
            ChunkedResultType &currentNPFB1 = std::get<1>(res);

            if (std::get<2>(currentFB1) > std::get<2>(bestDevFB1)) {
                bestDevFB1 = currentFB1;
            }
            if (std::get<2>(currentNPFB1) > std::get<2>(bestDevNPFB1)) {
                bestDevNPFB1 = currentNPFB1;
            }
            auto sf = std::cerr.flags();
            auto sp = std::cerr.precision();
            std::cerr.flags(std::ios::fixed);
            std::cerr.precision(2);
            std::cerr << "current iteration FB1-score  : " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentFB1) << "/" << std::get<1>(currentFB1) << "/" << std::get<2>(currentFB1) << "\t best FB1-score  : " << std::get<0>(bestDevFB1) << "/" << std::get<1>(bestDevFB1) << "/" << std::get<2>(bestDevFB1) << std::endl;
            std::cerr << "current iteration NPFB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentNPFB1) << "/" << std::get<1>(currentNPFB1) << "/" << std::get<2>(currentNPFB1) << "\t best NPFB1-score: " << std::get<0>(bestDevNPFB1)  << "/" << std::get<1>(bestDevNPFB1) << "/" << std::get<2>(bestDevNPFB1) << std::endl;
            std::cerr.flags(sf);
            std::cerr.precision(sp);
        }

        start = clock();

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<XPU> batchCumulatedGrads(num_in, num_hidden, num_out, m_featEmbManagerPtr, sstream, false);
        // begin to multi thread Training
#pragma omp parallel num_threads(CConfig::nThread)
        {
            int threadIndex = omp_get_thread_num();
            // int threadIndex = 0;
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            Model<XPU> cumulatedGrads(num_in, num_hidden, num_out, m_featEmbManagerPtr, sstream, false);

            // for evary instance in this mini-batch
            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                // fetch a to-be-trained instance
                GlobalExample *example = currentThreadData[inst];

                TNNets tnnets(m_nBeamSize, num_in, num_hidden, num_out, &modelParas, netss[threadIndex]);

                // decode and update
                // std::cerr << "begin to decode!" << std::endl;
                BeamDecoder decoder(&(example->instance), 
                                    m_transSystemPtr,
                                    m_featManagerPtr,
                                    m_featEmbManagerPtr,
                                    m_nBeamSize, 
                                    lattices[threadIndex],
                                    lattice_indexes[threadIndex],
                                    true);

                State * predState = decoder.decode(tnnets, example);

                tnnets.updateTNNetParas(&cumulatedGrads, decoder.beam, decoder.bEarlyUpdate, decoder.nGoldTransitionIndex, decoder.goldScoredTran);
            } // end for instance traverse

#pragma omp barrier
#pragma omp critical
            batchCumulatedGrads.mergeModel(&cumulatedGrads);

        } // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        end = clock();
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double time_used = (double)(end - start) / CLOCKS_PER_SEC;

            std::cerr << "[" << iter << "] totally train " << batch_size << " sentences, \ttime: " << time_used << " \taverage: " << batch_size / time_used << " sentences/second!" << std::endl; 
        }
    } // end total iteration

    for (int i = 0; i < CConfig::nThread; i++) {
        for (int j = 0; j < netss[i].size(); j++) {
            delete netss[i][j];
        }
    }
    for (int i = 0; i < CConfig::nThread; i++) {
        delete []lattices[i];
        delete []lattice_indexes[i];
    }
    DeleteStream(sstream);
    ShutdownTensorEngine<XPU>();
}

void BeamChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void BeamChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_dictManagerPtr->init(goldSet);

    m_featManagerPtr.reset(new FeatureManager());
    m_featManagerPtr->init(goldSet, m_dictManagerPtr);

    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager(
        m_featManagerPtr,
                static_cast<real_t>(CConfig::fInitRange)
                ));

    m_transSystemPtr.reset(new ActionStandardSystem());
    m_transSystemPtr->init(goldSet);

    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    std::cerr << std::endl << "  train set size: " << trainSet.size() << std::endl;
}
