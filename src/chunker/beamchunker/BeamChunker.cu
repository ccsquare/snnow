#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BeamChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <chrono>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "TNNets.h"
#include "BeamChunker.h"
#include "BatchBeamDecoder.h"
#include "Evalb.h"

#include "Example.h"

#include "BeamChunkerThread.h"

const double MICROSECOND = 1000000.0;

BeamChunker::BeamChunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
}

BeamChunker::BeamChunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
}

BeamChunker::~BeamChunker() {
}
    
std::pair<BeamChunker::ChunkedResultType, BeamChunker::ChunkedResultType> BeamChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<cpu> &modelParas) {
    int threads_num = CConfig::nThread;

    static int chunkRound = 1;

    auto start = std::chrono::high_resolution_clock::now();

    ChunkedDataSet predictDevSet(goldDevSet.size());

    std::vector<ChunkedDataSet> threadPredictDevSets(threads_num);

#pragma omp parallel num_threads(threads_num)
    {
        int threadIndex = omp_get_thread_num();
        SetDevice<XPU>(threadIndex);

        m_chunkerThreadPtrs[threadIndex]->chunk(threads_num, modelParas, devInstances, threadPredictDevSets[threadIndex]);
    }

    for (int i = 0; i < threads_num; i++) {
        for (int j = 0; j < threadPredictDevSets[i].size(); j++) {
            predictDevSet[i + j * threads_num] = threadPredictDevSets[i][j];
        }
    }

    auto end = std::chrono::high_resolution_clock::now();

    double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / MICROSECOND;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, \ttime: " << time_used << " \taverage: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    auto FB1 = Evalb::eval(predictDevSet, goldDevSet);
    auto NPFB1 = Evalb::eval(predictDevSet, goldDevSet, true);

    return std::make_pair(FB1, NPFB1);
}

void BeamChunker::generateMultiThreadsMiniBatchData(std::vector<std::vector<GlobalExample *>> &multiThread_miniBatch_data) {
    std::random_shuffle(gExamples.begin(), gExamples.end());

    // prepare mini-batch data for each threads
    static int exampleNumOfThread = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()))/ CConfig::nThread;

    auto sp = gExamples.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        std::vector<GlobalExample *> threadExamples;
        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(&(*p));
        }
        sp = ep;
        ep += exampleNumOfThread;
        multiThread_miniBatch_data.push_back(threadExamples);
    }
}

void BeamChunker::initBeamChunkerThread(InstanceSet &devSet) {
    const int trainLongestLen = (*std::max_element(gExamples.begin(), 
                gExamples.end(), 
                [](GlobalExample &ge1, GlobalExample &ge2) { 
                    return ge1.instance.input.size() < ge2.instance.input.size();
                })
            ).instance.input.size();
    static int devLongestLen = (*std::max_element(devSet.begin(), 
                devSet.end(), 
                [](Instance &inst1, Instance &inst2){ 
                    return inst1.input.size() < inst2.input.size(); 
                })
            ).input.size();

    const int longestLen = std::max(trainLongestLen, devLongestLen);
    std::cerr << "  longest sentence size: " << longestLen << std::endl;

    m_chunkerThreadPtrs.resize(CConfig::nThread);
    for (int i = 0; i < CConfig::nThread; i++) {
        m_chunkerThreadPtrs[i].reset(new BeamChunkerThread(i, m_nBeamSize, *(m_modelPtr.get()), m_transSystemPtr, m_featManagerPtr, m_featEmbManagerPtr, longestLen));
    }
}

void BeamChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {

    std::cerr << "[train involved]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[dev involved]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    std::cerr << "[chunkthreads involved]Initing chunkerthreads ..." << std::endl;
    initBeamChunkerThread(devSet);

    Model<cpu> &modelParas = *(m_modelPtr.get());
    auto featureTypes = m_featManagerPtr->getFeatureTypes();
    Model<cpu> adaGradSquares(num_in, num_hidden, num_out, featureTypes, NULL);

    ChunkedResultType bestDevFB1 = std::make_tuple(0.0, 0.0, -1.0);
    ChunkedResultType bestDevNPFB1 = std::make_tuple(0.0, 0.0, -1.0);

    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (CConfig::saveModel && iter % CConfig::nSaveModelPerIters == 0) {
            saveChunker(iter);
        }
        if (iter % CConfig::nEvaluatePerIters == 0) {
            auto res = chunk(devSet, devGoldSet, modelParas);
            ChunkedResultType &currentFB1 = std::get<0>(res);
            ChunkedResultType &currentNPFB1 = std::get<1>(res);

            if (std::get<2>(currentFB1) > std::get<2>(bestDevFB1)) {
                bestDevFB1 = currentFB1;
                bestDevNPFB1 = currentNPFB1;
                if (CConfig::saveModel) {
                    saveChunker(0);
                }
            }
            auto sf = std::cerr.flags();
            auto sp = std::cerr.precision();
            std::cerr.flags(std::ios::fixed);
            std::cerr.precision(2);
            std::cerr << "current iteration FB1-score  : " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentFB1) << "/" << std::get<1>(currentFB1) << "/" << std::get<2>(currentFB1) << "\t best FB1-score  : " << std::get<0>(bestDevFB1) << "/" << std::get<1>(bestDevFB1) << "/" << std::get<2>(bestDevFB1) << std::endl;
            std::cerr << "current iteration NPFB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << std::get<0>(currentNPFB1) << "/" << std::get<1>(currentNPFB1) << "/" << std::get<2>(currentNPFB1) << "\t best NPFB1-score: " << std::get<0>(bestDevNPFB1)  << "/" << std::get<1>(bestDevNPFB1) << "/" << std::get<2>(bestDevNPFB1) << std::endl;
            std::cerr.flags(sf);
            std::cerr.precision(sp);
        }

        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container, and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<cpu> batchCumulatedGrads(num_in, num_hidden, num_out, featureTypes, NULL);

        // begin to multi thread Training
#pragma omp parallel num_threads(CConfig::nThread)
        {
            int threadIndex = omp_get_thread_num();
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            Model<cpu> cumulatedGrads(num_in, num_hidden, num_out, featureTypes, NULL);

            SetDevice<gpu>(threadIndex);
            m_chunkerThreadPtrs[threadIndex]->train(modelParas, currentThreadData, cumulatedGrads);

#pragma omp barrier
#pragma omp critical
            batchCumulatedGrads.mergeModel(&cumulatedGrads);

        } // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        auto end = std::chrono::high_resolution_clock::now();
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / MICROSECOND;

            static const int batch_size = std::min(CConfig::nBeamBatchSize, static_cast<int>(gExamples.size()));
            std::cerr << "[" << iter << "] totally train " << batch_size << " sentences, \ttime: " << time_used << " \taverage: " << batch_size / time_used << " sentences/second!" << std::endl; 
        }
    } // end total iteration

}

void BeamChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void BeamChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_featManagerPtr.reset(new FeatureManager());
    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager());
    m_transSystemPtr.reset(new ActionStandardSystem());
    if (CConfig::loadModel){
        std::ifstream dict_is(CConfig::strModelDirPath + "/dictionarymanager.model");
        m_dictManagerPtr->loadDictManager(dict_is);

        std::ifstream featManager_is(CConfig::strModelDirPath + "/featuremanager.model");
        m_featManagerPtr->loadFeatureManager(featManager_is, m_dictManagerPtr);

        std::ifstream trans_is(CConfig::strModelDirPath + "/actionsystem.model");
        m_transSystemPtr->loadActionSystem(trans_is);
    } else {
        m_dictManagerPtr->init(goldSet);
        m_featManagerPtr->init(goldSet, m_dictManagerPtr);
        m_transSystemPtr->init(goldSet);
    }

    m_featEmbManagerPtr->init(m_featManagerPtr);

    num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    num_hidden = CConfig::nHiddenSize;
    num_out = m_transSystemPtr->getActNumber();

    srand(0);

    // Stream<XPU> *sstream = NewStream<XPU>();

    m_modelPtr.reset(new Model<cpu>(num_in, num_hidden, num_out, m_featEmbManagerPtr->getFeatureTypes(), NULL));
    if (CConfig::loadModel) {
        std::ifstream model_is(CConfig::strModelDirPath + "/netmodel.model");
        m_modelPtr->loadModel(model_is);
    } else {
        m_modelPtr->randomInitialize();
    }

    if (!CConfig::loadModel && CConfig::bReadPretrain) {
        m_featEmbManagerPtr->readPretrainedEmbeddings(*(m_modelPtr.get()));
    }

    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    auto featureTypes = m_featManagerPtr->getFeatureTypes();

    std::cerr << "  total input embedding dim: " << m_featEmbManagerPtr->getTotalFeatEmbSize() << std::endl;
    std::cerr << std::endl << "  train set size: " << trainSet.size() << std::endl;
    std::cerr << "  [begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "    " << ft.typeName << ":" << std::endl;
        std::cerr << "      dictSize = " << ft.dictSize << std::endl;
        std::cerr << "      featSize = " << ft.featSize << std::endl;
        std::cerr << "      embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "  [end]" << std::endl;
}

void BeamChunker::saveChunker(int round) {
    std::string dir = CConfig::strModelDirPath;
    std::string app_str;

    if (round != -1) {
        app_str = "." + std::to_string(round);
    }

    std::ofstream actionSystemOs(dir + "/actionsystem.model" + app_str);
    m_transSystemPtr->saveActionSystem(actionSystemOs);

    std::ofstream dictOs(dir + "/dictionarymanager.model" + app_str);
    m_dictManagerPtr->saveDictManager(dictOs);

    std::ofstream featManagerOs(dir + "/featuremanager.model" + app_str);
    m_featManagerPtr->saveFeatureManager(featManagerOs);

    std::ofstream modelOs(dir + "/netmodel.model" + app_str);
    m_modelPtr->saveModel(modelOs);
}
