#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BatchBeamDecoder.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Sat 16 Jan 2016 03:10:51 PM CST
 ************************************************************************/
#include "TNNets.h"
#include "BatchBeamDecoder.h"

BatchBeamDecoderMemoryManager::BatchBeamDecoderMemoryManager(const int beamSize, const int decoderItemSize, const int longestLen, const int threadNum) :
    m_nThread(threadNum),
    m_nBeamSize(beamSize),
    m_nDecoderItemSize(decoderItemSize),
    lattices(threadNum, std::vector<State *>(decoderItemSize)),
    lattice_indexes(threadNum, std::vector<State **>(decoderItemSize))
{
    const int nMaxLatticeSize = (beamSize + 1) * longestLen;

    for (int i = 0; i < m_nThread; i++) {
        for (int j = 0; j < m_nDecoderItemSize; j++) {
            lattices[i][j] = new State[nMaxLatticeSize];
            lattice_indexes[i][j] = new State*[longestLen + 2];
        }
    }
}

BatchBeamDecoderMemoryManager::~BatchBeamDecoderMemoryManager() {
    for (int i = 0; i < m_nThread; i++) {
        for (int j = 0; j < m_nDecoderItemSize; j++) {
            delete []lattices[i][j];
            delete []lattice_indexes[i][j];
        }
    }
}

std::vector<State *> BatchBeamDecoder::decode(TNNets &tnnet, std::vector<GlobalExample *> &gExamplePtrs) {
    assert (m_bTrain);

    std::vector<State *> retvals(m_nInstSize, nullptr);

    for (int i = 0; i < m_nInstSize; i++) {
        State *s = m_lLatticePtrs[i];
        const int nMaxLatticeSize = m_lnMaxLatticeSizes[i];
        const int nSentLen = m_lnSentLens[i];

        for (int j = 0; j < nMaxLatticeSize; j++) {
            s[j].sentLength = nSentLen;
        }
    }

    for (int i = 0; i < m_nInstSize; i++) {
        m_lLatticePtrs[i][0].setBeamIdx(0);
        m_lLatticePtrs[i][0].clear();

        m_lLattice_indexPtrs[i][0] = m_lLatticePtrs[i];
        m_lLattice_indexPtrs[i][1] = m_lLattice_indexPtrs[i][0] + 1;
    }

    const int batch_size = m_nInstSize * m_nBeamSize;
    const int num_in = tnnet.num_in;
    const int num_out = tnnet.num_out;

    TensorContainer<cpu, 2, real_t> input(Shape2(batch_size, num_in));
    TensorContainer<cpu, 2, real_t> pred(Shape2(batch_size, num_out));

    std::vector<bool> itemCompeleteds(m_nInstSize, false);
    auto checkCompleted = [&itemCompeleteds]() -> bool {
        for (auto b : itemCompeleteds) {
            if (!b) {
                return false;
            }
        }

        return true;
    };
    int nRound = 1;
    while (!checkCompleted()) {
        std::vector<FeatureVector> batchFeatureVectors;
        tnnet.moveToNextNet();
        for (int insti = 0; insti < m_nInstSize; insti++) {
            if (!itemCompeleteds[insti]) {
                m_lnExpandRounds[insti] = nRound;
            }
        }
        generateBatchInput(num_in, nRound, tnnet, itemCompeleteds, input, batchFeatureVectors);
        tnnet.addFeatVecs(batchFeatureVectors);

        pred = 0.0;
        // batch forward compute 
        tnnet.Forward(input, pred);

        generateBeams(pred, nRound, gExamplePtrs, itemCompeleteds);

        lazyExpandBeams(nRound, itemCompeleteds, retvals);

        // prepare lattice for next chunking round
        for (int insti = 0; insti < m_nInstSize; insti++) {
            if (itemCompeleteds[insti]) {
                continue;
            }

            m_lLattice_indexPtrs[insti][nRound + 1] = m_lLattice_indexPtrs[insti][nRound] + m_lBeamPtrs[insti]->currentBeamSize;
        }

        // check each brach if it is completed
        for (int insti = 0; insti < m_nInstSize; insti++) {
            if (nRound >= m_lnMaxRounds[insti]) {
                itemCompeleteds[insti] = true;
            }
        }

        nRound++;
    }

    return retvals;
}

// void generateBeamExamples(std::vector<BeamExample> &beamExamples) {

// }

// void generateBeamExamplesOf(std::vector<BeamExample> &beamExamples, int instId) {
//     Beam &beam = *(m_lBeamPtrs[instId].get());
//     bool earlyUpdate = m_lbEarlyUpdates[instId];
//     int goldTransitIndex = m_lnGoldTransitionIndex[instId];
//     CScoredTransition &goldTransit = m_lGoldScoredTrans[instId];

//     real_t sum = 0.0;
//     real_t maxScore = beam.getMaxScoreInBeam();

//     std::vector<CScoredTransition *> trainingData;

//     for (int bi = 0; bi < beam.currentBeamSize; bi++) {
//         trainingData.push_back(beam.beam + bi);
//     }

//     if (earlyUpdate) {
//         trainingData.push_back(&goldTransit);
//         goldTransitIndex = static_cast<int>(trainingData.size()) - 1;
//     }

//     std::vector<real_t> updateParas(trainingData.size(), 0.0);

//     for (int bi = 0; bi < static_cast<int>(trainingData.size()); bi++) {
//         updateParas[bi] = exp(trainingData[bi]->score - maxScore);
//         sum += updateParas[bi];
//     }
//     for (int bi = 0; bi < static_cast<int>(trainingData.size()); bi++) {
//         updateParas[bi] = updateParas[bi] / sum;
//     }
//     updateParas[goldTransitIndex] -= 1.0;

//     // for (int backRound = m_lnExpandRounds[instId] - 1; backRound >= 0; --backRound) {
//     //     TensorContainer<cpu, 2, real_t> 
//     // }
// }

void BatchBeamDecoder::generateBatchInput(const int num_in, const int nRound, const TNNets &tnnet, const std::vector<bool> &itemCompeleteds, TensorContainer<cpu, 2, real_t> &input, std::vector<FeatureVector> &batchFeatureVectors) {
    input = 0.0;

    int input_index = 0;
    TensorContainer<cpu, 2, real_t> branch_input(Shape2(m_nBeamSize, num_in));
    // fill full in the batch input
    for (int insti = 0; insti < m_nInstSize; insti++, input_index += m_nBeamSize) {
        if (itemCompeleteds[insti]) {
            continue;
        }

        branch_input = 0.0;
        std::vector<FeatureVector> featureVectors;
        Beam &beam = *(m_lBeamPtrs[insti].get());
        State *state = m_lLattice_indexPtrs[insti][nRound - 1];
        Instance *inst = m_lInstPtrss[insti];

        int curBeamSize = nRound == 1 ? 1 : beam.currentBeamSize;
        featureVectors.resize(curBeamSize);
        generateBatchInputForState(state, inst, featureVectors);
        for (int i = 0; i < curBeamSize; i++) {
            batchFeatureVectors.push_back(featureVectors[i]);
        }
        for (int i = curBeamSize; i < m_nBeamSize; i++) {
            batchFeatureVectors.push_back(FeatureVector ());
        }

        m_featEmbManagerPtr->returnInput(featureVectors, tnnet.modelParas->featEmbs, branch_input);

        for (int beami = 0; beami < curBeamSize; beami++) {
            for (int num_ini = 0; num_ini < num_in; num_ini++) {
                // TODO: to be accelerated
                input[input_index + beami][num_ini] = branch_input[beami][num_ini];
            }
        }
    }
}

void BatchBeamDecoder::generateBeams(const TensorContainer<cpu, 2, real_t> &pred, const int nRound, std::vector<GlobalExample *> &gExamplePtrs, std::vector<bool> &itemCompeleteds) {
    for (int insti = 0; insti < m_nInstSize; insti++) {
        if (itemCompeleteds[insti]) {
            continue;
        }

        Beam &beam = *(m_lBeamPtrs[insti].get());
        GlobalExample *gExample = gExamplePtrs[insti];

        beam.clear();

        State *start_state = m_lLattice_indexPtrs[insti][nRound - 1];
        State *end_state   = m_lLattice_indexPtrs[insti][nRound];
        int stateIdx = 0;
        for (State *currentState = start_state; currentState != end_state; ++currentState, ++stateIdx) {
            std::vector<int> validActs;
            m_transSystemPtr->generateValidActs(*currentState, validActs);

            bool noValid = true;
            for (unsigned actId = 0; actId < static_cast<int>(validActs.size()); ++actId) {
                if (validActs[actId] == -1) {
                    continue;
                }

                noValid = false;
                CScoredTransition trans(currentState, actId, currentState->score + pred[stateIdx][actId]);

                if (currentState->bGold && actId == gExample->goldActs[nRound - 1]) {
                    trans.bGold = true;
                    m_lGoldScoredTrans[insti] = trans;
                }

                beam.insert(trans);
            }
            assert (noValid == false);
        }

        m_lbEarlyUpdates[insti] = true;
        for (int beami = 0; beami < m_nBeamSize; beami++) {
            if (beam.beam[beami].bGold) {
                m_lbEarlyUpdates[insti] = false;
            }
        }

        if (m_lbEarlyUpdates[insti]) {
            itemCompeleteds[insti] = true;
        }
    }
}

void BatchBeamDecoder::lazyExpandBeams(const int nRound, const std::vector<bool> &itemCompeleteds, std::vector<State *> &retvals) {
    for (int insti = 0; insti < m_nInstSize; insti++) {
        if (itemCompeleteds[insti]) {
            continue;
        }

        real_t dMaxScore = 0.0;

        Beam &beam = *(m_lBeamPtrs[insti].get());
        State **lattice_index = m_lLattice_indexPtrs[insti];
        int &nGoldTransitionIndex = m_lnGoldTransitionIndex[insti];
        State *&retval = retvals[insti];

        for (int beami = 0; beami < beam.currentBeamSize; beami++) {
            const CScoredTransition &transition = beam.beam[beami];

            State *target = lattice_index[nRound] + beami;
            *target = *(transition.source);
            m_transSystemPtr->move(*(transition.source), *target, transition);

            target->bGold = transition.bGold;
            target->setBeamIdx(beami);
            if (transition.bGold) {
                nGoldTransitionIndex = beami;
            }

            if (beami == 0 || target->score > dMaxScore) {
                dMaxScore = target->score;
                retval = target;
            }
        }
    }
}

void BatchBeamDecoder::generateBatchInputForBeam(std::vector<State *> &statePtrs, std::vector<Instance *> &instPtrs, std::vector<std::vector<FeatureVector>> &featVecVecs) {
    for (int insti = 0; insti < static_cast<int>(featVecVecs.size()); insti++) {
        std::vector<FeatureVector> &featVecs = featVecVecs[insti];
        State *state = statePtrs[insti];
        Instance *inst = instPtrs[insti];

        for (int i = 0; i < static_cast<int>(featVecs.size()); i++) {
            m_featManagerPtr->extractFeature(*(state + i), *inst, featVecs[i]);
        }
    }
}

void BatchBeamDecoder::generateBatchInputForState(State *state, Instance *inst, std::vector<FeatureVector> &featVecs) {
    for (int i = 0; i < static_cast<int>(featVecs.size()); i++) {
        m_featManagerPtr->extractFeature(*(state + i), *inst, featVecs[i]);
    }
}
