#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: TNNets.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Sat 16 Jan 2016 03:07:15 PM CST
 ************************************************************************/

#include "TNNets.h"
#include "BatchBeamDecoder.h"
#include "BeamDecoder.h"

TNNetsMemoryManager::TNNetsMemoryManager(const int threadNum, const int longestLen, const int batchSize, const int num_in, const int num_hidden, const int num_out, Model<XPU> *modelParasPtr) : m_nThread(threadNum) {
    for (int i = 0; i < m_nThread; i++) {
        std::vector<NNet<XPU> *> nets(longestLen + 1);

        for (int j = 0; j < nets.size(); j++) {
            nets[j] = new NNet<XPU>(batchSize, num_in, num_hidden, num_out, modelParasPtr);
        }

        netss.push_back(nets);
    }
}

TNNetsMemoryManager::~TNNetsMemoryManager() {
    for (int i = 0; i < m_nThread; i++) {
        for (int j = 0; j < netss[i].size(); j++) {
            delete netss[i][j];
        }
    }
}

std::vector<NNet<XPU> *> TNNetsMemoryManager::getNetPtrVec(const int threadId) {
    return netss[threadId];
}

/*
 * computes the gradients of beam contrastive learning
 */
void TNNets::updateTNNetParas(Model<XPU> *cumulatedGrads, BeamDecoder &decoder) {
    Beam &beam = decoder.beam;
    bool earlyUpdate = decoder.bEarlyUpdate;
    int goldTransitIndex = decoder.nGoldTransitionIndex;
    CScoredTransition &goldTransit = decoder.goldScoredTran;
   
    float sum =0;
    float maxScore = beam.getMaxScoreInBeam();

    // TODO: predict correctly ?
    if (!earlyUpdate && beam.isMaxScoreGold()) {
        return ;
    }
    /*
     * construct the training data
     */
    std::vector<CScoredTransition*> trainingData;

    for(int bi = 0; bi < beam.currentBeamSize; bi++)
        trainingData.push_back( beam.beam + bi );
    if( earlyUpdate ){
        trainingData.push_back( & goldTransit );
        goldTransitIndex = trainingData.size() - 1;
    }

    std::vector<double> updateParas(trainingData.size(), 0); // updating parameter vector

    /*
     * get gradients with beam contrastive learning
     * sentence-level loglikelihood and softmax
     */
    for (int b_j = 0; b_j < trainingData.size(); b_j++) { // for every transit in the beam
        updateParas[b_j] = exp( trainingData[b_j]->score - maxScore );
        sum += updateParas[b_j];
    }
    for (int b_j = 0; b_j < trainingData.size(); b_j++)
        updateParas[b_j] = updateParas[b_j] / sum;
    updateParas[ goldTransitIndex ] -= 1.0;

    /*  
     *  Back propagation updating
     *  from last parsing state to the former states
     */
    for(int backRound = netIdx - 1; backRound >= 0; --backRound){
        //std::cout<<"backRound:\t"<<backRound<<std::endl;
        TensorContainer<cpu, 2, real_t> grads;
        grads.Resize( Shape2( batch_size, num_out ) );
        grads = 0.0;
        int i = 0;
        for(auto iter = trainingData.begin(); iter != trainingData.end(); iter++, i++){
            grads[ ( *iter )->source->beamIdx ][ ( *iter )->action ] += updateParas[i] / CConfig::nBeamBatchSize;
            if( backRound != 0 ){ // last time updating, do not need to prepare for next iteration
                 ( *iter )->action = ( *iter )->source->lastAction;
                 ( *iter )->source = ( *iter )->source->prevStatePtr;
            }
        }

        // FOR CHECK
        nets[backRound]->Backprop(grads);

        nets[backRound]->SubsideGradsTo(cumulatedGrads, netFeatVecs[backRound]);
    }
}

void TNNets::updateTNNetParas(Model<XPU> *cumulatedGrads, BatchBeamDecoder &batchDecoder) {
    const int itemSize = batchDecoder.m_nInstSize;
    std::vector<real_t> maxScores(itemSize, 0.0);
    std::vector<std::vector<real_t>> updateParasVec(itemSize);

    std::vector<std::vector<CScoredTransition *>> trainingDatas(itemSize);
    std::vector<bool> predictCorrect(itemSize, false);

    for (int insti = 0; insti < itemSize; insti++) {
        real_t sum = 0.0;
        real_t &maxScore = maxScores[insti];

        Beam &beam = *(batchDecoder.m_lBeamPtrs[insti].get());

        maxScore = beam.getMaxScoreInBeam();

        bool earlyUpdate = batchDecoder.m_lbEarlyUpdates[insti];

        if (!earlyUpdate && beam.isMaxScoreGold()) {
            predictCorrect[insti] = true;
        }

        int &goldTransitIndex = batchDecoder.m_lnGoldTransitionIndex[insti];
        CScoredTransition &goldTransit = batchDecoder.m_lGoldScoredTrans[insti];
        std::vector<CScoredTransition *> &trainingData = trainingDatas[insti];

        for (int beami = 0; beami < beam.currentBeamSize; beami++) {
            trainingData.push_back(beam.beam + beami);
        }
        if (earlyUpdate) {
            trainingData.push_back(&goldTransit);
            goldTransitIndex = trainingData.size() - 1;
        }

        updateParasVec[insti] = std::vector<real_t>(trainingData.size(), 0.0);

        std::vector<real_t> &updateParas = updateParasVec[insti];
        for (int beami = 0; beami < static_cast<int>(trainingData.size()); beami++) {
            updateParas[beami] = exp(trainingData[beami]->score - maxScore);
            sum += updateParas[beami];
        }
        for (int beami = 0; beami < static_cast<int>(trainingData.size()); beami++) {
            updateParas[beami] = updateParas[beami] / sum;
        }
        updateParas[goldTransitIndex] -= 1.0;
    }

    for (int backRound = netIdx - 1; backRound >= 0; --backRound) {
        TensorContainer<cpu, 2, real_t> grads(Shape2(batch_size, num_out), static_cast<real_t>(0.0));

        int baseIndex = 0;
        for (int insti = 0; insti < itemSize; insti++, baseIndex += batchDecoder.m_nBeamSize) {
            if (predictCorrect[insti] || batchDecoder.m_lnExpandRounds[insti] < backRound + 1) {
                continue;
            }
           
            std::vector<CScoredTransition *> &trainingData = trainingDatas[insti];
            std::vector<real_t> &updateParas = updateParasVec[insti];
            int i = 0;
            for (auto iter = trainingData.begin(); iter != trainingData.end(); iter++, i++) {
                grads[(*iter)->source->beamIdx + baseIndex][(*iter)->action] += updateParas[i] / CConfig::nBeamBatchSize;

                if (backRound != 0) {
                    (*iter)->action = (*iter)->source->lastAction;
                    (*iter)->source = (*iter)->source->prevStatePtr;
                }
            }
        }

        // FOR CKECK
        nets[backRound]->Backprop(grads);
        nets[backRound]->SubsideGradsTo(cumulatedGrads, netFeatVecs[backRound]);
    }
}
