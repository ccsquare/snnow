#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: BeamDecoder.cu
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include "TNNets.h"
#include "mshadow/tensor.h"

#include "BeamDecoder.h"

BeamDecoder::BeamDecoder(Instance *inst, 
            std::shared_ptr<ActionStandardSystem> transitionSystemPtr, 
            std::shared_ptr<FeatureManager> featureMangerPtr,
            std::shared_ptr<FeatureEmbeddingManager> featureEmbManagerPtr, 
            int beamSize, 
            bool bTrain) : 
            m_transSystemPtr(transitionSystemPtr),
            m_featManagerPtr(featureMangerPtr),
            m_featEmbManagerPtr(featureEmbManagerPtr),
            beam(beamSize) {
    nSentLen = inst->input.size();
    nMaxRound = nSentLen;

    nMaxLatticeSize = (beamSize + 1) * nMaxRound;
    nRound = 0;

    this->inst = inst;
    this->bTrain = bTrain;

    bEarlyUpdate = false;
   
    lattice = new State[nMaxLatticeSize];
    lattice_index = new State *[nMaxRound + 2];
}

BeamDecoder::BeamDecoder(Instance *inst, 
            std::shared_ptr<ActionStandardSystem> transitionSystemPtr, 
            std::shared_ptr<FeatureManager> featureMangerPtr,
            std::shared_ptr<FeatureEmbeddingManager> featureEmbManagerPtr, 
            int beamSize, 
            State *lattice,
            State **lattice_index,
            bool bTrain) : 
            m_transSystemPtr(transitionSystemPtr),
            m_featManagerPtr(featureMangerPtr),
            m_featEmbManagerPtr(featureEmbManagerPtr),
            beam(beamSize) {
    nSentLen = inst->input.size();
    nMaxRound = nSentLen;

    nMaxLatticeSize = (beamSize + 1) * nMaxRound;
    nRound = 0;

    this->inst = inst;
    this->bTrain = bTrain;

    bEarlyUpdate = false;
   
    this->lattice = lattice;
    this->lattice_index = lattice_index; 
    // lattice = new State[nMaxLatticeSize];
    // lattice_index = new State *[nMaxRound + 2];
}

BeamDecoder::~BeamDecoder() {
    if (!bTrain) {
        delete []lattice;
        delete []lattice_index;
    }
}

void BeamDecoder::generateLabeledSequence(TNNets &tnnets, LabeledSequence &predictedSent) {

    State *predState = decode(tnnets);

    m_transSystemPtr->generateOutput(*predState, predictedSent);
}

State* BeamDecoder::decode(TNNets &tnnet, GlobalExample *gExample) {
    State *retval = nullptr;

    for (int i = 0; i < nMaxLatticeSize; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    if (bTrain) {
        // to know which neural net this state is generated from,
        // which is used for batch updating in the end of training
        lattice[0].setBeamIdx(0); 
    }

    lattice[0].clear();

    //  set  lattice_index as point to states in lattice
    lattice_index[0] = lattice;
    lattice_index[1] = lattice_index[0] + 1;

    // input layer and predicted layer for each round
    TensorContainer<cpu, 2, real_t> input;
    input.Resize(Shape2(beam.beamFullSize, tnnet.num_in));

    TensorContainer<cpu, 2, real_t> pred;
    pred.Resize(Shape2(beam.beamFullSize, tnnet.num_out));

    // for every round in training
    for (nRound = 1; nRound <= nMaxRound; nRound++) {
        input = 0.0;
        pred = 0.0;

        // In the training process, we need a new neural net to forward, 
        // with which, we can directly update parameters in the end other
        // than forwarding and updating!
        if (bTrain) {
            tnnet.genNextStepNet();
        }

        // extract features and generate input embeddings
        // std::vector<std::vector<int>> featureVectors; // extracted feature vectors in batch.
        std::vector<FeatureVector> featureVectors;
        featureVectors.resize(nRound == 1 ? 1 : beam.currentBeamSize);
        generateInputBatch(lattice_index[nRound - 1], inst, featureVectors);
        if (bTrain) {
            tnnet.addFeatVecs(featureVectors);
        }
        m_featEmbManagerPtr->returnInput(featureVectors, tnnet.modelParas->featEmbs, input);

        tnnet.Forward(input, pred);

        // clear the beam for the next beam expand
        beam.clear();

        // for each state in the latest beam, expand it and insert expanded state into next beam
        int stateIdx = 0;
        for (State *currentState = lattice_index[nRound - 1]; currentState != lattice_index[nRound]; ++currentState, ++stateIdx) {
            std::vector<int> validActs;
            m_transSystemPtr->generateValidActs(*currentState, validActs);

            bool noValid = true;
            // for each valid action
            for (unsigned actId = 0; actId < validActs.size(); ++actId) {
                // skip invalid action
                if (validActs[actId] == -1) {
                    continue;
                }

                noValid = false;
                // construct scored transition and insert it into beam
                CScoredTransition trans(currentState, actId, currentState->score + pred[stateIdx][actId]); // TODO: ignore inValid scores ?
                
                // if this is the gold transition
                if (bTrain && currentState->bGold && actId == gExample->goldActs[nRound - 1]) {
                    trans.bGold = true;
                    goldScoredTran = trans;
                }
                beam.insert(trans);
            }
            assert (noValid == false);
        }

        bEarlyUpdate = true; // early update if gold transition was not inserted into the beam
        for (int i = 0; i < beam.currentBeamSize; ++i) {
            if (beam.beam[i].bGold) {
                bEarlyUpdate = false;
            }
        }

        if (bTrain && bEarlyUpdate) {
            break;
        }

        float dMaxScore = 0.0;
        // lazy expand the target states in the beam
        for (int i = 0; i < beam.currentBeamSize; ++i) {
            const CScoredTransition transition = beam.beam[i];

            State *target = lattice_index[nRound] + i;
            *target = *(transition.source);
            m_transSystemPtr->move(*(transition.source), *target, transition);

            if (bTrain) {
                target->bGold = transition.bGold;
                target->setBeamIdx(i);       // the corresponding nnet to be forwarded in the tnnets of specific round
                if (transition.bGold) {
                    nGoldTransitionIndex = i;
                }
            }

            if (i == 0 || target->score > dMaxScore) {
                dMaxScore = target->score;
                retval = target;
            }
        }

        // prepare lattice for next chunking round
        lattice_index[nRound + 1] = lattice_index[nRound] + beam.currentBeamSize;
    }

    return retval; // return without early updating
}
