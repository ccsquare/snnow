#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunkerThread.cu
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Sat 23 Jan 2016 03:30:55 PM CST
 ************************************************************************/
#include "GreedyChunkerThread.h"

GreedyChunkerThread::GreedyChunkerThread(
        const int threadId, 
        const int batchSize, 
        Model<cpu> &paraModel, 
        std::shared_ptr<ActionStandardSystem> transitionSystemPtr, 
        std::shared_ptr<FeatureManager> featureMangerPtr,
        std::shared_ptr<FeatureEmbeddingManager> featureEmbManagerPtr, 
        int longestLen) : 
        m_transSystemPtr(transitionSystemPtr),
        m_featManagerPtr(featureMangerPtr),
        m_featEmbManagerPtr(featureEmbManagerPtr),
        m_nThreadId(threadId), 
        m_nBatchSize(batchSize)
{
    m_nNumIn = paraModel.Wi2h.shape_[0];
    m_nNumHidden = paraModel.Wi2h.shape_[1];
    m_nNumOut = paraModel.Wh2o.shape_[1];

    InitTensorEngine<gpu>(threadId);

    stream = NewStream<gpu>();

    modelPtr.reset(new Model<gpu>(m_nNumIn, m_nNumHidden, m_nNumOut, paraModel.featTypes, stream, false, m_nThreadId));
    modelPtr->featEmbs = paraModel.featEmbs;
    modelPtr->featTypes = paraModel.featTypes;

    statePtr = new State[longestLen + 1];
}

GreedyChunkerThread::~GreedyChunkerThread() {
    DeleteStream(stream);

    ShutdownTensorEngine<gpu>();

    delete []statePtr;
}

void GreedyChunkerThread::train(Model<cpu> &paraModel, std::vector<Example *> &examplePtrs, const int miniBatchSize, Model<cpu> &cumulatedGrads, int &threadCorrectSize, double &threadObjLoss) {
    // copy from the parameter model to current model
    Copy(modelPtr->Wi2h, paraModel.Wi2h, stream);
    Copy(modelPtr->Wh2o, paraModel.Wh2o, stream);
    Copy(modelPtr->hbias, paraModel.hbias, stream);

    Model<gpu> grads(m_nNumIn, m_nNumHidden, m_nNumOut, modelPtr->featTypes, stream);

    std::shared_ptr<NNet<gpu>> nnet(new NNet<gpu>(m_nBatchSize, m_nNumIn, m_nNumHidden, m_nNumOut, modelPtr.get()));

    std::vector<FeatureVector> featureVectors(m_nBatchSize);
    TensorContainer<cpu, 2, real_t> input(Shape2(m_nBatchSize, m_nNumIn));

    std::vector<std::vector<int>> validActsVec(m_nBatchSize);
    TensorContainer<cpu, 2, real_t> pred(Shape2(m_nBatchSize, m_nNumOut));

    for (unsigned inst = 0; inst < static_cast<int>(examplePtrs.size()); inst += m_nBatchSize) {
        input = 0.0;
        pred  = 0.0;
        for (unsigned insti = 0; (insti < m_nBatchSize) && (inst + insti < static_cast<int>(examplePtrs.size())); insti++) {
            Example *e = examplePtrs[inst + insti];

            featureVectors[insti] = e->features;
            validActsVec[insti] = e->labels;
        }
        m_featEmbManagerPtr->returnInput(featureVectors, modelPtr->featEmbs, input);

        nnet->Forward(input, pred, CConfig::bDropOut);

        for (unsigned insti = 0; (insti < m_nBatchSize) && (inst + insti < static_cast<int>(examplePtrs.size())); insti++) {
            int optAct = -1;
            int goldAct = -1;

            std::vector<int> &validActs = validActsVec[insti];
            for (int i = 0; i < validActs.size(); i++) {
                if (validActs[i] >= 0) {
                    if (optAct == -1 || pred[insti][i] > pred[insti][optAct]){
                        optAct = i;
                    }

                    if (validActs[i] == 1) {
                        goldAct = i;
                    }
                }
            }
            if (optAct == goldAct) {
                threadCorrectSize += 1;
            }

            real_t maxScore = pred[insti][optAct];
            real_t goldScore = pred[insti][goldAct];

            real_t sum = 0.0;
            for (int i = 0; i < validActs.size(); i++) {
                if (validActs[i] >= 0) {
                    pred[insti][i] = std::exp(pred[insti][i] - maxScore);
                    sum += pred[insti][i];
                }
            }

            threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / miniBatchSize;

            for (int i = 0; i < validActs.size(); i++) {
                if (validActs[i] >= 0) {
                    pred[insti][i] = pred[insti][i] / sum;
                } else {
                    pred[insti][i] = 0.0;
                }
            }
            pred[insti][goldAct] -= 1.0;
        }

        pred /= static_cast<real_t>(miniBatchSize);

        nnet->Backprop(pred);
        nnet->SubsideGradsTo(&grads, featureVectors);
    }

    // copy grads from current grads to cumulatedGrads
    Copy(cumulatedGrads.Wi2h, grads.Wi2h, stream);
    Copy(cumulatedGrads.Wh2o, grads.Wh2o, stream);
    Copy(cumulatedGrads.hbias, grads.hbias, stream);
    for (int i = 0; i < static_cast<int>(cumulatedGrads.featEmbs.size()); i++) {
        cumulatedGrads.featEmbs[i] = grads.featEmbs[i];
    }
}

void GreedyChunkerThread::chunk(const int threads_num, Model<cpu> &paraModel, InstanceSet &devInstances, ChunkedDataSet &labeledSents) {
    Copy(modelPtr->Wi2h, paraModel.Wi2h, stream);
    Copy(modelPtr->Wh2o, paraModel.Wh2o, stream);
    Copy(modelPtr->hbias, paraModel.hbias, stream);

    for (unsigned inst = m_nThreadId; inst < static_cast<unsigned>(devInstances.size()); inst += threads_num) {
        LabeledSequence predictSent(devInstances[inst].input);

        State* predState = decode(&(devInstances[inst]));

        m_transSystemPtr->generateOutput(*predState, predictSent);

        labeledSents.push_back(predictSent);
    }
}

State* GreedyChunkerThread::decode(Instance *inst) {
    Model<gpu> &modelParas = *(modelPtr.get());

    State *lattice = statePtr;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transSystemPtr.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, m_nNumIn, m_nNumHidden, m_nNumOut, &modelParas));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].sentLength = nSentLen;
    }

    lattice[0].clear();

    TensorContainer<cpu, 2, real_t> input(Shape2(1, m_nNumIn));
    TensorContainer<cpu, 2, real_t> pred(Shape2(1, m_nNumOut));
       
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        input = 0.0;
        pred = 0.0;

        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        std::vector<FeatureVector> featureVectors(1);
        // featureVectors[0].clear();
        generateInputBatch(currentState, inst, featureVectors);
        m_featEmbManagerPtr->returnInput(featureVectors, modelParas.featEmbs, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
        
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
    }

    return retval;
}

void GreedyChunkerThread::generateInputBatch(State *state, Instance *inst, std::vector<FeatureVector> &featvecs) {
    for (int i = 0; i < featvecs.size(); i++) {
        m_featManagerPtr->extractFeature(*(state + i), *inst, featvecs[i]);
    }
}
