#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <chrono>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#include "GreedyChunkerThread.h"

const double MICROSECOND = 1000000.0;

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() { } 

std::pair<GreedyChunker::ChunkedResultType, GreedyChunker::ChunkedResultType> GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<cpu> &modelParas) {
    int threads_num = CConfig::nThread;

    static int chunkRound = 1;

    auto start = std::chrono::high_resolution_clock::now();

    ChunkedDataSet predictDevSet(goldDevSet.size());

    std::vector<ChunkedDataSet> threadPredictDevSets(threads_num);

#pragma omp parallel num_threads(threads_num)
    {
        int threadIndex =  omp_get_thread_num();
        SetDevice<gpu>(threadIndex);

        m_chunkerThreadPtrs[threadIndex]->chunk(threads_num, modelParas, devInstances, threadPredictDevSets[threadIndex]);
#pragma omp barrier
    }

    for (int i = 0; i < threads_num; i++) {
        for (int j = 0; j < static_cast<int>(threadPredictDevSets[i].size()); j++) {
            predictDevSet[i + j * threads_num] = threadPredictDevSets[i][j];
        }
    }

    auto end = std::chrono::high_resolution_clock::now();

    double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / MICROSECOND;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    ChunkedResultType FB1 = Evalb::eval(predictDevSet, goldDevSet);
    ChunkedResultType NPFB1 = Evalb::eval(predictDevSet, goldDevSet, true);

    return std::make_pair(FB1, NPFB1);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, Model<cpu> &modelParas, double batchObjLoss, double posClassificationRate, ChunkedResultType &bestDevFB1, ChunkedResultType &bestDevNPFB1) {
    auto res = chunk(devSet, devGoldSet, modelParas);

    ChunkedResultType &currentFB1 = std::get<0>(res);
    ChunkedResultType &currentNPFB1 = std::get<1>(res);
    if (std::get<2>(currentFB1) > std::get<2>(bestDevFB1)) {
        bestDevFB1 = currentFB1;
        bestDevNPFB1 = currentNPFB1;
        if (CConfig::saveModel) {
            saveChunker(0);
        }
    }

    double loss = batchObjLoss;

    auto sf = std::cerr.flags();
    auto sp = std::cerr.precision();
    std::cerr.flags(std::ios::fixed);
    std::cerr.precision(2);
    std::cerr << "current iteration FB1-score  : " << std::get<0>(currentFB1) << "/" << std::get<1>(currentFB1) << "/" << std::get<2>(currentFB1) << "\tbest FB1-score  : " << std::get<0>(bestDevFB1) << "/" << std::get<1>(bestDevFB1) << "/" << std::get<2>(bestDevFB1) << std::endl;
    std::cerr << "current iteration NPFB1-score: " << std::get<0>(currentNPFB1) << "/" << std::get<1>(currentNPFB1) << "/" << std::get<2>(currentNPFB1) << "\tbest NPFB1-score: " << std::get<0>(bestDevNPFB1) << "/" << std::get<1>(bestDevNPFB1) << "/" << std::get<2>(bestDevNPFB1) << std::endl;
    std::cerr << "current objective fun-score  : " << loss << "\tclassfication rate: " << posClassificationRate << std::endl;
    std::cerr.flags(sf);
    std::cerr.precision(sp);
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());

    static int exampleNumOfThread = std::min(CConfig::nGreedyBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void GreedyChunker::saveChunker(int round) {
    std::string dir = CConfig::strModelDirPath;
    std::string app_str;

    if (round != -1) {
        app_str = "." + std::to_string(round);
    }

    std::ofstream actionSystemOs(dir + "/actionsystem.model" + app_str);
    m_transSystemPtr->saveActionSystem(actionSystemOs);

    std::ofstream dictOs(dir + "/dictionarymanager.model" + app_str);
    m_dictManagerPtr->saveDictManager(dictOs);

    std::ofstream featManagerOs(dir + "/featuremanager.model" + app_str);
    m_featManagerPtr->saveFeatureManager(featManagerOs);

    std::ofstream modelOs(dir + "/netmodel.model" + app_str);
    m_modelPtr->saveModel(modelOs);
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "[train involved]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[dev involved]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    std::cerr << "[chunkthreads involved]Initing chunkerthreads..." << std::endl;
    initGreedyChunkerThread(devSet);

    Model<cpu> &modelParas = *(m_modelPtr.get());
    auto featureTypes = m_featManagerPtr->getFeatureTypes();
    Model<cpu> adaGradSquares(num_in, num_hidden, num_out, featureTypes, NULL);

    ChunkedResultType bestDevFB1 = std::make_tuple(0.0, 0.0, -1.0);
    ChunkedResultType bestDevNPFB1 = std::make_tuple(0.0, 0.0, -1.0);

    const int batchSize = std::min(CConfig::nGreedyBatchSize, static_cast<int>(trainExamplePtrs.size()));
    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (CConfig::saveModel && iter % CConfig::nSaveModelPerIters == 0) {
            saveChunker(iter);
        }
        if (iter % CConfig::nEvaluatePerIters == 0) {

            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            double regular_loss = 0.5 * CConfig::fRegularizationRate * modelParas.norm2();
            printEvaluationInfor(devSet, devGoldSet, modelParas, batchObjLoss + regular_loss, posClassificationRate, bestDevFB1, bestDevNPFB1);
            // std::cerr << "Regularization loss: " << regular_loss << std::endl;
            // printEvaluationInfor(devSet, devGoldSet, modelParas, batchObjLoss, posClassificationRate, bestDevFB1, bestDevNPFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<cpu> batchCumulatedGrads(num_in, num_hidden, num_out, featureTypes, NULL);
        
#pragma omp parallel num_threads(CConfig::nThread)
        {
            int threadIndex = omp_get_thread_num();
            SetDevice<gpu>(threadIndex);

            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;

            Model<cpu> cumulatedGrads(num_in, num_hidden, num_out, featureTypes, NULL);

            auto currentThreadData = multiThread_miniBatch_data[threadIndex];
            m_chunkerThreadPtrs[threadIndex]->train(modelParas, currentThreadData, batchSize, cumulatedGrads, threadCorrectSize, threadObjLoss);

#pragma omp barrier
#pragma omp critical 
            batchCumulatedGrads.mergeModel(&cumulatedGrads);

#pragma omp critical 
            batchCorrectSize += threadCorrectSize;
#pragma omp critical 
            batchObjLoss += threadObjLoss;
        
        }  // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        auto end = std::chrono::high_resolution_clock::now();
        if (iter % CConfig::nEvaluatePerIters == 0) 
        {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / MICROSECOND;
            std::cerr << "[" << iter << "] totally train " << batchSize << " examples, time: " << time_used << " average: " << batchSize / time_used << " examples/second!" << std::endl; 
        }
    }
}

void GreedyChunker::initGreedyChunkerThread(InstanceSet &devSet) {
    const int longestLen = (*std::max_element(devSet.begin(), 
                devSet.end(), 
                [](Instance &inst1, Instance &inst2) { 
                    return inst1.size() < inst2.size();
                })
            ).input.size();

    std::cerr << "  longest sentence size: " << longestLen << std::endl;
    
    m_chunkerThreadPtrs.resize(CConfig::nThread);
    for (int i = 0; i < CConfig::nThread; i++) {
        m_chunkerThreadPtrs[i].reset(new GreedyChunkerThread(i, CConfig::nGPUBatchSize, *(m_modelPtr.get()), m_transSystemPtr, m_featManagerPtr, m_featEmbManagerPtr, longestLen));
    }
}

void GreedyChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_featManagerPtr.reset(new FeatureManager());
    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager());
    m_transSystemPtr.reset(new ActionStandardSystem());
    if (CConfig::loadModel){
        std::ifstream dict_is(CConfig::strModelDirPath + "/dictionarymanager.model");
        m_dictManagerPtr->loadDictManager(dict_is);

        std::ifstream featManager_is(CConfig::strModelDirPath + "/featuremanager.model");
        m_featManagerPtr->loadFeatureManager(featManager_is, m_dictManagerPtr);

        std::ifstream trans_is(CConfig::strModelDirPath + "/actionsystem.model");
        m_transSystemPtr->loadActionSystem(trans_is);
    } else {
        m_dictManagerPtr->init(goldSet);
        m_featManagerPtr->init(goldSet, m_dictManagerPtr);
        m_transSystemPtr->init(goldSet);
    }

    m_featEmbManagerPtr->init(m_featManagerPtr);

    num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    num_hidden = CConfig::nHiddenSize;
    num_out = m_transSystemPtr->getActNumber();

    srand(0);

    m_modelPtr.reset(new Model<cpu>(num_in, num_hidden, num_out, m_featEmbManagerPtr->getFeatureTypes(), NULL));
    if (CConfig::loadModel) {
        std::ifstream model_is(CConfig::strModelDirPath + "/netmodel.model");
        m_modelPtr->loadModel(model_is);
    } else {
        m_modelPtr->randomInitialize();
    }

    if (!CConfig::loadModel && CConfig::bReadPretrain) {
        m_featEmbManagerPtr->readPretrainedEmbeddings(*(m_modelPtr.get()));
    }

    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }

    auto featureTypes = m_featManagerPtr->getFeatureTypes();

    std::cerr << "  total input embedding dim: " << m_featEmbManagerPtr->getTotalFeatEmbSize() << std::endl;
    std::cerr << "  greedy train set size: " << trainExamplePtrs.size() << std::endl;
    std::cerr << "  [begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "    " << ft.typeName << ":" << std::endl;
        std::cerr << "      dictSize = " << ft.dictSize << std::endl;
        std::cerr << "      featSize = " << ft.featSize << std::endl;
        std::cerr << "      embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "  [end]" << std::endl;
}

