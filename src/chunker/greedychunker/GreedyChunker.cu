#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() { } 

std::pair<GreedyChunker::ChunkedResultType, GreedyChunker::ChunkedResultType> GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, Model<XPU> &modelParas) {
    static int chunkRound = 1;
    static auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
    std::vector<State *> lattices(CConfig::nThread);
    for (int i = 0; i < lattices.size(); i++) {
        lattices[i] = new State[longestInst.size() + 1];
    }

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(LabeledSequence(currentInstance.input));
    }

#pragma omp parallel num_threads(CConfig::nThread)
    {
        int threadIndex =  omp_get_thread_num();

        for (unsigned inst = threadIndex; inst < devInstances.size(); inst += CConfig::nThread) {
            Instance &currentInstance = devInstances[inst];

            State* predState = decode(&currentInstance, modelParas, lattices[threadIndex]);

            LabeledSequence &predSent = predDevSet[inst];

            m_transSystemPtr->generateOutput(*predState, predSent);
        }
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "[" << chunkRound << "] totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl; chunkRound++;

    for (int i = 0; i< lattices.size(); i++) {
        delete []lattices[i];
    }

    ChunkedResultType FB1 = Evalb::eval(predDevSet, goldDevSet);
    ChunkedResultType NPFB1 = Evalb::eval(predDevSet, goldDevSet, true);

    return std::make_pair(FB1, NPFB1);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, Model<XPU> &modelParas, double batchObjLoss, double posClassificationRate, ChunkedResultType &bestDevFB1, ChunkedResultType &bestDevNPFB1) {
    auto res = chunk(devSet, devGoldSet, modelParas);

    ChunkedResultType &currentFB1 = std::get<0>(res);
    ChunkedResultType &currentNPFB1 = std::get<1>(res);
    if (std::get<2>(currentFB1) > std::get<2>(bestDevFB1)) {
        bestDevFB1 = currentFB1;
    }
    if (std::get<2>(currentNPFB1) > std::get<2>(bestDevNPFB1)) {
        bestDevNPFB1 = currentNPFB1;
    }

    double loss = batchObjLoss;

    auto sf = std::cerr.flags();
    auto sp = std::cerr.precision();
    std::cerr.flags(std::ios::fixed);
    std::cerr.precision(2);
    std::cerr << "current iteration FB1-score  : " << std::get<0>(currentFB1) << "/" << std::get<1>(currentFB1) << "/" << std::get<2>(currentFB1) << "\tbest FB1-score  : " << std::get<0>(bestDevFB1) << "/" << std::get<1>(bestDevFB1) << "/" << std::get<2>(bestDevFB1) << std::endl;
    std::cerr << "current iteration NPFB1-score: " << std::get<0>(currentNPFB1) << "/" << std::get<1>(currentNPFB1) << "/" << std::get<2>(currentNPFB1) << "\tbest NPFB1-score: " << std::get<0>(bestDevNPFB1) << "/" << std::get<1>(bestDevNPFB1) << "/" << std::get<2>(bestDevNPFB1) << std::endl;
    std::cerr << "current objective fun-score  : " << loss << "\tclassfication rate: " << posClassificationRate << std::endl;
    std::cerr.flags(sf);
    std::cerr.precision(sp);
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());

    static int exampleNumOfThread = std::min(CConfig::nGreedyBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void display1Tensor( Tensor<XPU, 1, real_t> & tensor ){
    for(int i = 0; i < tensor.size(0); i++)
        std::cerr<<tensor[i]<<" ";
    std::cerr<<std::endl;
}

void display2Tensor( Tensor<XPU, 2, double> tensor ){
    std::cerr<<"size 0 :" << tensor.size(0)<<" size 1: "<<tensor.size(1)<<std::endl;
    for(int i = 0; i < tensor.size(0); i++){
       for(int j = 0; j < tensor.size(1); j++)
           std::cerr<<tensor[i][j]<<" ";
       std::cerr<<std::endl;
    }
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "[trainingSet involved initing]Initing DictManager &  FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "[devSet involved initing]Initing generateInstanceSetCache for devSet..." << std::endl;
    initDev(devSet);

    const static int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transSystemPtr->getActNumber();
    const static int batchSize = std::min(CConfig::nGreedyBatchSize, static_cast<int>(trainExamplePtrs.size()));
    // const static int batchSize = static_cast<int>(trainExamplePtrs.size());

    srand(0);

    Stream<XPU> *sstream = NewStream<XPU>();
    InitTensorEngine<XPU>();

    auto featureTypes = m_featManagerPtr->getFeatureTypes();

    std::cerr << "[begin]featureTypes:" << std::endl;
    for (auto &ft : featureTypes) {
        std::cerr << "  " << ft.typeName << ":" << std::endl;
        std::cerr << "    dictSize = " << ft.dictSize << std::endl;
        std::cerr << "    featSize = " << ft.featSize << std::endl;
        std::cerr << "    embsSize = " << ft.featEmbSize << std::endl;
    }
    std::cerr << "[end]" << std::endl;

    Model<XPU> modelParas(num_in, num_hidden, num_out, featureTypes, sstream, true);
    // m_featEmbManagerPtr->readPretrainedEmbeddings(modelParas);

    Model<XPU> adaGradSquares(num_in, num_hidden, num_out, featureTypes, sstream, false);

    ChunkedResultType bestDevFB1 = std::make_tuple(0.0, 0.0, -1.0);
    ChunkedResultType bestDevNPFB1 = std::make_tuple(0.0, 0.0, -1.0);

    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;

    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            // std::cerr << "2-norm: " << modelParas.norm2() << std::endl;
            // std::cerr << "2-embedding-norm: " << modelParas.embeddings_norm2() << std::endl;
            printEvaluationInfor(devSet, devGoldSet, modelParas, batchObjLoss + 0.5 * CConfig::fRegularizationRate * modelParas.norm2(), posClassificationRate, bestDevFB1, bestDevNPFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        clock_t start, end;
        start = clock();

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

        Model<XPU> batchCumulatedGrads(num_in, num_hidden, num_out, featureTypes, sstream, false);
        
// #pragma omp parallel num_threads(CConfig::nThread)
        {
            int threadIndex = omp_get_thread_num();
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;

            Model<XPU> cumulatedGrads(num_in, num_hidden, num_out, featureTypes, sstream, false);
            std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(CConfig::nGPUBatchSize, num_in, num_hidden, num_out, &modelParas));

            std::vector<FeatureVector> featureVectors(CConfig::nGPUBatchSize);
            TensorContainer<cpu, 2, real_t> input(Shape2(CConfig::nGPUBatchSize, num_in));

            std::vector<std::vector<int>> validActsVec(CConfig::nGPUBatchSize);
            TensorContainer<cpu, 2, real_t> pred(Shape2(CConfig::nGPUBatchSize, num_out));

            for (unsigned inst = 0; inst < currentThreadData.size(); inst += static_cast<unsigned>(CConfig::nGPUBatchSize)) {
                input = 0.0;
                pred  = 0.0;
                for (unsigned insti = 0; insti < static_cast<unsigned >(CConfig::nGPUBatchSize); insti++) {
                    Example *e = currentThreadData[inst + insti];

                    featureVectors[insti] = e->features;
                    validActsVec[insti] = e->labels;
                }
                m_featEmbManagerPtr->returnInput(featureVectors, modelParas.featEmbs, input);

                nnet->Forward(input, pred, false);

                for (unsigned insti = 0; insti < static_cast<unsigned>(CConfig::nGPUBatchSize); insti++) {
                    int optAct = -1;
                    int goldAct = -1;

                    std::vector<int> &validActs = validActsVec[insti];
                    for (int i = 0; i < validActs.size(); i++) {
                        if (validActs[i] >= 0) {
                            if (optAct == -1 || pred[insti][i] > pred[insti][optAct]){
                                optAct = i;
                            }

                            if (validActs[i] == 1) {
                                goldAct = i;
                            }
                        }
                    }
                    if (optAct == goldAct) {
                        threadCorrectSize += 1;
                    }

                    real_t maxScore = pred[insti][optAct];
                    real_t goldScore = pred[insti][goldAct];

                    real_t sum = 0.0;
                    for (int i = 0; i < validActs.size(); i++) {
                        if (validActs[i] >= 0) {
                            pred[insti][i] = std::exp(pred[insti][i] - maxScore);
                            sum += pred[insti][i];
                        }
                    }

                    threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;

                    for (int i = 0; i < validActs.size(); i++) {
                        if (validActs[i] >= 0) {
                            pred[insti][i] = pred[insti][i] / sum;
                        } else {
                            pred[insti][i] = 0.0;
                        }
                    }
                    pred[insti][goldAct] -= 1.0;
                }

                pred /= static_cast<real_t>(batchSize);

                nnet->Backprop(pred);
                nnet->SubsideGradsTo(&cumulatedGrads, featureVectors);
            }

// #pragma omp barrier
// #pragma omp critical 
            batchCumulatedGrads.mergeModel(&cumulatedGrads);

// #pragma omp critical 
            batchCorrectSize += threadCorrectSize;

// #pragma omp critical 
            batchObjLoss += threadObjLoss;
        
        }  // end multi-processor

        modelParas.update(&batchCumulatedGrads, &adaGradSquares);

        end = clock();
        if (iter % CConfig::nEvaluatePerIters == 0) 
        {
            double time_used = (double)(end - start) / CLOCKS_PER_SEC;
            std::cerr << "[" << iter << "] totally train " << batchSize << " examples, time: " << time_used << " average: " << batchSize / time_used << " examples/second!" << std::endl; 
        }
    }

    ShutdownTensorEngine<XPU>();
}

void GreedyChunker::initDev(InstanceSet &devSet) {
    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), devSet);
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_dictManagerPtr.reset(new DictManager());
    m_dictManagerPtr->init(goldSet);

    m_featManagerPtr.reset(new FeatureManager());
    m_featManagerPtr->init(goldSet, m_dictManagerPtr);

    m_featEmbManagerPtr.reset(new FeatureEmbeddingManager(
                m_featManagerPtr->getFeatureTypes(),
                m_featManagerPtr->getDictManagerPtrs(),
                static_cast<real_t>(CConfig::fInitRange)));

    std::cerr << "  total input embedding dim: " << m_featEmbManagerPtr->getTotalFeatEmbSize() << std::endl;
    m_transSystemPtr.reset(new ActionStandardSystem());
    m_transSystemPtr->init(goldSet);

    Instance::generateInstanceSetCache(*(m_dictManagerPtr.get()), trainSet);

    GlobalExample::generateTrainingExamples(*(m_transSystemPtr.get()), *(m_featManagerPtr.get()), trainSet, goldSet, gExamples);

    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }

    std::cerr << "  Greedy train set size: " << trainExamplePtrs.size() << std::endl;
}

State* GreedyChunker::decode(Instance *inst, Model<XPU> &modelParas, State *lattice) {
    const static int num_in = m_featEmbManagerPtr->getTotalFeatEmbSize();
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transSystemPtr->getActNumber();

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transSystemPtr.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &modelParas));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

    TensorContainer<cpu, 2, real_t> input(Shape2(1, num_in));
    TensorContainer<cpu, 2, real_t> pred(Shape2(1, num_out));
       
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        input = 0.0;
        pred = 0.0;

        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        std::vector<FeatureVector> featureVectors(1);
        // featureVectors[0].clear();
        generateInputBatch(currentState, inst, featureVectors);
        m_featEmbManagerPtr->returnInput(featureVectors, modelParas.featEmbs, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
        
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
    }

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<FeatureVector> &featvecs) {
    for (int i = 0; i < featvecs.size(); i++) {
        m_featManagerPtr->extractFeature(*(state + i), *inst, featvecs[i]);
    }
}
