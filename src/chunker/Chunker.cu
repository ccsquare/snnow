#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: Chunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Chunker.h"
#include "TNNets.h"
#include "Evalb.h"

#define DEBUG

#ifdef DEBUG
// #define DEBUG1
// #define DEBUG2
// #define DEBUG4
// #define DEBUG5
#define CHECKNETVALUES
#endif

Chunker::Chunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
    m_bEarlyUpdate = false;
}

Chunker::Chunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
    m_bEarlyUpdate = false;
}

Chunker::~Chunker() {

}
    
double Chunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {
    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;
    const int beam_size = CConfig::nBeamSize;

    TNNets tnnets(beam_size, num_in, num_hidden, num_out, &netsParas, false);

    clock_t start, end;
    start = clock();
    ChunkedDataSet predictDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        ChunkedSentence predictSent(devInstances[inst].input);

        BeamDecoder decoder(&(devInstances[inst]), 
                            m_transitionSystem.get(),
                            m_featExtractor.get(),
                            m_fEmb.get(),
                            m_nBeamSize, 
                            false);

        decoder.generateChunkedSentence(tnnets, predictSent);

        predictDevSet.push_back(predictSent);
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    auto res = Evalb::eval(predictDevSet, goldDevSet);

    return std::get<2>(res);
}

void Chunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    initTrain(trainGoldSet, trainSet);

    m_featExtractor->generateInstanceSetCache(devSet);

    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;
    const int batch_size = std::min(CConfig::nBatchSize, static_cast<int>(gExamples.size()));

    const int beam_size = CConfig::nBeamSize;

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(beam_size, num_in, num_hidden, num_out);

    double bestDevFB1 = -1.0;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double currentFB1 = chunk(devSet, devGoldSet, netsParas);
            if (currentFB1 > bestDevFB1) {
                bestDevFB1 = currentFB1;
            }
            auto sf = std::cerr.flags();
            auto sp = std::cerr.precision();
            std::cerr.flags(std::ios::fixed);
            std::cerr.precision(2);
            std::cerr << "current iteration FB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentFB1 << "\t best FB1-score: " << bestDevFB1 << std::endl;
            std::cerr.flags(sf);
            std::cerr.precision(sp);
        }

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;

        // prepare mini-batch data for each threads
        std::random_shuffle(gExamples.begin(), gExamples.end());
        int exampleNumOfThread = batch_size / CConfig::nThread;
        auto sp = gExamples.begin();
        auto ep = sp + exampleNumOfThread;
        for (int i = 0; i < CConfig::nThread; i++) {
            std::vector<GlobalExample *> threadExamples;
            for (auto p = sp; p != ep; p++) {
                threadExamples.push_back(&(*p));
            }
            sp = ep;
            ep += exampleNumOfThread;
            multiThread_miniBatch_data.push_back(threadExamples);
        }

        UpdateGrads<XPU> batchCumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
        // begin to multi thread Training
#pragma omp parallel
        {
            auto currentThreadData = multiThread_miniBatch_data[omp_get_thread_num()];
            UpdateGrads<XPU> threadCumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);

            // for evary instance in this mini-batch
            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                // fetch a to-be-trained instance
                GlobalExample *example = currentThreadData[inst];

                TNNets tnnets(m_nBeamSize, num_in, num_hidden, num_out, &netsParas);

                // decode and update
                // std::cerr << "begin to decode!" << std::endl;
                BeamDecoder decoder(&(example->instance), 
                                    m_transitionSystem.get(),
                                    m_featExtractor.get(),
                                    m_fEmb.get(),
                                    m_nBeamSize, 
                                    true);

                State * predState = decoder.decode(tnnets, example);

                tnnets.updateTNNetParas(threadCumulatedGrads, decoder.beam, decoder.bEarlyUpdate, decoder.nGoldTransitionIndex, decoder.goldScoredTran);
            } // end for instance traverse

#pragma omp barrier
#pragma omp critical
            {
                batchCumulatedGrads.cg_hbias = batchCumulatedGrads.cg_hbias + threadCumulatedGrads.cg_hbias;
                batchCumulatedGrads.cg_Wi2h = batchCumulatedGrads.cg_Wi2h + threadCumulatedGrads.cg_Wi2h;
                batchCumulatedGrads.cg_Wh2o = batchCumulatedGrads.cg_Wh2o + threadCumulatedGrads.cg_Wh2o;
            }
        } // end multi-processor
        NNet<XPU>::UpdateCumulateGrads(batchCumulatedGrads, &netsParas);
    } // end total iteration
}
    
void Chunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    cerr << "Training init..." << endl;
    cerr << "  Training Instance num: " << trainSet.size() << endl;

    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

#ifdef DEBUG
    m_transitionSystem->displayLabel2ActionIdx();
#endif

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            m_nBeamSize));

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

#ifdef DEBUG1
    std::cerr << "train set size: " << trainSet.size() << std::endl;
    std::cerr << "dev gold set size: " << goldSet.size() << std::endl;
    std::cerr << "global examples size: " << gExamples.size() << std::endl;
#endif
}
