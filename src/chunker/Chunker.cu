#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: Chunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Chunker.h"
#include "TNNets.h"
#include "Evalb.h"

#define DEBUG

#ifdef DEBUG
// #define DEBUG1
// #define DEBUG2
#define DEBUG4
#define DEBUG5
#endif

Chunker::Chunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
    m_bEarlyUpdate = false;
}

Chunker::Chunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
    m_bEarlyUpdate = false;
}

Chunker::~Chunker() {

}
    
double Chunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {
    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;
    const int beam_size = CConfig::nBeamSize;

    TNNets tnnets(beam_size, num_in, num_hidden, num_out, &netsParas, false);

    // ChunkedDataSet predSents(devInstances.size());

    clock_t start, end;
    start = clock();
    ChunkedDataSet predictDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        // std::cout << "inst = " << inst << std::endl;
        // char tch;
        // std::cin >> tch;
        // predSents[inst].init(devInstances[inst].input);

        ChunkedSentence predictSent(devInstances[inst].input);

        BeamDecoder decoder(&(devInstances[inst]), 
                            m_transitionSystem.get(),
                            m_featExtractor.get(),
                            m_fEmb.get(),
                            m_nBeamSize, 
                            false);

        decoder.generateChunkedSentence(tnnets, predictSent);
#ifdef DEBUG1
        std::cout << "predictSent's size: " << predictSent.m_lChunkedWords.size() << std::endl;
#endif
        //State *predState = new State();
        // m_transitionSystem->generateOutput(*predState, predictSent);

        predictDevSet.push_back(predictSent);
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    auto res = Evalb::eval(predictDevSet, goldDevSet);

    return std::get<2>(res);
}

void Chunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
#ifdef DEBUG2
    char ch;
#endif 
    initTrain(trainGoldSet, trainSet);

    m_featExtractor->generateInstanceSetCache(devSet);

    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;

    const int beam_size = CConfig::nBeamSize;

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(beam_size, num_in, num_hidden, num_out);
    for (int ii = 0; ii < netsParas.Wi2h.shape_[0]; ii++) {
        for (int jj = 0; jj < netsParas.Wi2h.shape_[1]; jj++){
            if (netsParas.Wi2h[ii][jj]) {
                std::cout << "[Chunker train [1]: NaN appears in netsParas.Wi2h" << std::endl;
                char ch;
                std::cin >> ch;
            }
        }
    }
    double bestDevFB1 = -1.0;
#ifdef DEBUG2
    std::cout << "Before chunking..." << std::endl;
#endif 
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        // Evaluate FB1 score per iteration
        std::cout << "iter = " << iter << std::endl;
        if (iter == 2) {
            std::cout << "begin to debug" << std::endl;
        }
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double currentFB1 = chunk(devSet, devGoldSet, netsParas);
            if (currentFB1 > bestDevFB1) {
                bestDevFB1 = currentFB1;
            }
            std::cout << "current iteration FB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentFB1 << "\t best FB1-score: " << bestDevFB1 << std::endl;
        }

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;

        // prepare mini-batch data for each threads
        std::random_shuffle(gExamples.begin(), gExamples.end());
        int threadExampleNum = std::min(CConfig::nBatchSize, static_cast<int>(gExamples.size())) / CConfig::nThread;
        auto sp = gExamples.begin();
        auto ep = sp + threadExampleNum;
        for (int i = 0; i < CConfig::nThread; i++) {
            std::vector<GlobalExample *> threadExamples;
            for (auto p = sp; p != ep; p++) {
                threadExamples.push_back(&(*p));
            }
#ifdef DEBUG3
            std::cout << "threadExamples' size: " << threadExamples.size() << std::endl;
            std::cout << "globalExamples' size: " << gExamples.size() << std::endl;
#endif
            sp = ep;
            ep += threadExampleNum;
            multiThread_miniBatch_data.push_back(threadExamples);
        }

        // begin to multi thread Training
// #pragma omp parallel
        {
            auto currentThreadData = multiThread_miniBatch_data[omp_get_thread_num()];
            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);

            // for evary instance in this mini-batch
            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                // fetch a to-be-trained instance
                GlobalExample *example = currentThreadData[inst];

                TNNets tnnets(m_nBeamSize, num_in, num_hidden, num_out, &netsParas);
                for (int ii = 0; ii < netsParas.Wi2h.shape_[0]; ii++) {
                    for (int jj = 0; jj < netsParas.Wi2h.shape_[1]; jj++){
                        if (netsParas.Wi2h[ii][jj]) {
                            std::cout << "[Chunker train [2]: NaN appears in netsParas.Wi2h" << std::endl;
                            char ch;
                            std::cin >> ch;
                        }
                    }
                }

                // decode and update
                // std::cout << "begin to decode!" << std::endl;
                BeamDecoder decoder(&(example->instance), 
                                    m_transitionSystem.get(),
                                    m_featExtractor.get(),
                                    m_fEmb.get(),
                                    m_nBeamSize, 
                                    true);

                std::cout << "round: " << iter << "\tinst: " << inst << std::endl;

                State * predState = decoder.decode(tnnets, example);

                std::cout << "current beamsize: " << decoder.beam.currentBeamSize << std::endl;
                // std::cout << "end decoding!" << std::endl;

                // std::vector<int> predictedActions;
                // State *ptr = predState;
                // if (ptr == nullptr) {
                //     std::cout << "predstate is nullptr!" << std::endl;
                // }
                // int i = 1;
                // while (ptr != nullptr && ptr->last_action != -1) {
                //     i++;
                //     predictedActions.push_back(ptr->last_action);
                //     ptr = ptr->previous_;
                // }
                // std::cout << "decoded path length: " << i << std::endl;
                // std::cout << "[pred action sequences]: ";
                // for (int i = 0; i < predictedActions.size(); i++) {
                //     std::cout << predictedActions[predictedActions.size() - 1 - i] << " ";
                // }
                // std::cout << std::endl;
                // char tch;
                // std::cin >> tch;

                tnnets.updateTNNetParas(cumulatedGrads, decoder.beam, decoder.bEarlyUpdate, decoder.nGoldTransitionIndex, decoder.goldScoredTran);
                for (int ii = 0; ii < cumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wi2h.shape_[1]; jj++){
                        if (isnan(cumulatedGrads.cg_Wi2h[ii][jj])) {
                            std::cout << "W(input -> hidden): NaN appears!" << std::endl;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wh2o.shape_[1]; jj++){
                        if (isnan(cumulatedGrads.cg_Wh2o[ii][jj])) {
                            std::cout << "W(hidden -> output): NaN appears!" << std::endl;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_hbias.shape_[0]; ii++) {
                    if (isnan(cumulatedGrads.cg_hbias[ii])) {
                        std::cout << "Bias: NaN appears!" << std::endl;
                    }
                }
            } // end for instance traverse

// #pragma omp barrier
// #pragma omp critical

            NNet<XPU>::UpdateCumulateGrads(cumulatedGrads, &netsParas);
        } // end multi-processor
    } // end total iteration
}
    
void Chunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;

    cout << "Training init..." << endl;
    cout << "  Training Instance num: " << trainSet.size() << endl;

    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

#ifdef DEBUG
    m_transitionSystem->displayLabel2ActionIdx();
#endif

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            m_nBeamSize));

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

#ifdef DEBUG1
    std::cout << "train set size: " << trainSet.size() << std::endl;
    std::cout << "dev gold set size: " << goldSet.size() << std::endl;
    std::cout << "global examples size: " << gExamples.size() << std::endl;
#endif
}
