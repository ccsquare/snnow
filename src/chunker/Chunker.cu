#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: Chunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Thu 19 Nov 2015 03:59:17 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>

#include "Config.h"

#include "Beam.h"
#include "Chunker.h"
#include "TNNets.h"


Chunker::Chunker() {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = false;
    m_bEarlyUpdate = false;
}

Chunker::Chunker(bool isTrain) {
    m_nBeamSize = CConfig::nBeamSize;
    m_bTrain = isTrain;
    m_bEarlyUpdate = false;
}

Chunker::~Chunker() {

}
    
double Chunker::parse(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {
    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;
    const int beam_size = CConfig::nBeamSize;

    TNNets tnnets(beam_size, num_in, num_hidden, num_out, &netsParas, false);

    ChunkedDataSet predSents(devInstances.size());

    clock_t start, end;
    start = clock();
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        predSents[inst].init(devInstances[inst].input);
    }
    end = clock();



    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally parse " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    return 0.0;
}

void Chunker::train(ChunkedDataSet &goldSet, InstanceSet &trainSet, InstanceSet &devSet) {
    initTrain(goldSet, trainSet);

    m_featExtractor->generateInstanceSetCache(devSet);

    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = m_transitionSystem->nActNum;

    const int beam_size = CConfig::nBeamSize;

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(beam_size, num_in, num_hidden, num_out);
    double bestDevFB1 = -1.0;

    
}
    
void Chunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;

    cout << "Training init..." << endl;
    cout << "  Training Instance num: " << trainSet.size() << endl;

    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());
    m_transitionSystem->displayLabel2ActionIdx();

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            m_nBeamSize));

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);
}
