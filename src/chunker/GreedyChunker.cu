#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

#ifdef DEBUG
#define ADDREGURLOSS
// #define CLOSEOPENOMP
#endif

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() { } 

double GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {
    auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
    State *lattice = new State[longestInst.size() + 1];

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(ChunkedSentence(currentInstance.input));

        State* predState = decode(&currentInstance, netsParas, lattice);

        ChunkedSentence &predSent = predDevSet[inst];

        m_transitionSystem->generateOutput(*predState, predSent);
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    delete []lattice;

    auto res = Evalb::eval(predDevSet, goldDevSet);

    return std::get<2>(res);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, NNetPara<XPU> &netsPara, double batchObjLoss, double posClassificationRate, double &bestDevFB1) {
    double currentFB1 = chunk(devSet, devGoldSet, netsPara);
    if (currentFB1 > bestDevFB1) {
        bestDevFB1 = currentFB1;
    }

    double loss = batchObjLoss;
#ifdef ADDREGURLOSS
    double paraLoss = 0.0;
    for (int ii = 0; ii < netsPara.Wi2h.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wi2h.shape_[1]; jj++) {
            paraLoss += netsPara.Wi2h[ii][jj] * netsPara.Wi2h[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.Wh2o.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wh2o.shape_[1]; jj++) {
            paraLoss += netsPara.Wh2o[ii][jj] * netsPara.Wh2o[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.hbias.shape_[0]; ii++) {
        paraLoss += netsPara.hbias[ii] * netsPara.hbias[ii];
    }
    std::cout << "current |W|^2: " << paraLoss << std::endl;
    paraLoss *= 0.5 * CConfig::fRegularizationRate;

    loss += paraLoss;
#endif

    auto sf = std::cout.flags();
    auto sp = std::cout.precision();
    std::cout.flags(std::ios::fixed);
    std::cout.precision(2);
    std::cout << "current iteration FB1-score: " << currentFB1 << "\tbest FB1-score: " << bestDevFB1 << std::endl;
    std::cout << "current objective fun-score: " << loss << "\tclassfication rate: " << posClassificationRate << std::endl;
    std::cout.flags(sf);
    std::cout.precision(sp);
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    int exampleNumOfThread = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void display1Tensor( Tensor<cpu, 1, real_t> & tensor ){
    for(int i = 0; i < tensor.size(0); i++)
        std::cout<<tensor[i]<<" ";
    std::cout<<std::endl;
}

void display2Tensor( Tensor<cpu, 2, double> tensor ){
    std::cout<<"size 0 :" << tensor.size(0)<<" size 1: "<<tensor.size(1)<<std::endl;
    for(int i = 0; i < tensor.size(0); i++){
       for(int j = 0; j < tensor.size(1); j++)
           std::cout<<tensor[i][j]<<" ";
       std::cout<<std::endl;
    }
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cout << "Initing FeatureExtractor & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cout << "Excuting generateInstanceSetCache & readPretrainEmbeddings..." << std::endl;
    m_featExtractor->generateInstanceSetCache(devSet);

    // m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;
    const static int batchSize = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size()));

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    InitTensorEngine<XPU>();

    NNetPara<XPU> netsParas(1, num_in, num_hidden, num_out);

    double bestDevFB1 = -1.0;

    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;

    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            printEvaluationInfor(devSet, devGoldSet, netsParas, batchObjLoss, posClassificationRate, bestDevFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;

        // prepare mini-batch data for each threads
        // std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);
        UpdateGrads<XPU> batchCumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);


#ifndef CLOSEOPENOMP 
#pragma omp parallel
#endif
        {
            int threadIndex = omp_get_thread_num();
#ifndef CLOSEOPENOMP
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];
#endif
#ifdef CLOSEOPENOMP
            auto currentThreadData = multiThread_miniBatch_data[0];
#endif
            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;

            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
            std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));

            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {

                Example *e = currentThreadData[inst];

                TensorContainer<cpu, 2, real_t> input;
                input.Resize(Shape2(1, num_in));

                TensorContainer<cpu, 2, real_t> pred;
                pred.Resize(Shape2(1, num_out));

                std::vector<std::vector<int>> featureVectors;
                featureVectors.push_back(e->features);
                m_fEmb->returnInput(featureVectors, input);

                nnet->Forward(input, pred, false);

                std::vector<int> validActs(e->labels);

                int optAct = -1;
                int goldAct = -1;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        if (optAct == -1 || pred[0][i] > pred[0][optAct]){
                            optAct = i;
                        }

                        if (validActs[i] == 1) {
                            goldAct = i;
                        }
                    }
                }
                if (optAct == goldAct) {
                    threadCorrectSize += 1;
                }

                real_t maxScore = pred[0][optAct];
                real_t goldScore = pred[0][goldAct];

                real_t sum = 0.0;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = std::exp(pred[0][i] - maxScore);
                        sum += pred[0][i];
                    }
                }

                threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;

                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = pred[0][i] / sum;
                    } else {
                        pred[0][i] = 0.0;
                    }
                }
                pred[0][goldAct] -= 1.0;

                for (int i = 0; i < validActs.size(); i++) {
                    pred[0][i] /= batchSize;
                }

                nnet->Backprop(pred);
                nnet->SubsideGrads(cumulatedGrads);
            }

#ifndef CLOSEOPENOMP 
#pragma omp barrier
#pragma omp critical 
#endif            
            {
                batchCumulatedGrads.cg_hbias = batchCumulatedGrads.cg_hbias + cumulatedGrads.cg_hbias;
                batchCumulatedGrads.cg_Wi2h = batchCumulatedGrads.cg_Wi2h + cumulatedGrads.cg_Wi2h;
                batchCumulatedGrads.cg_Wh2o = batchCumulatedGrads.cg_Wh2o + cumulatedGrads.cg_Wh2o;
            }

#ifndef CLOSEOPENOMP 
#pragma omp critical 
#endif
            batchCorrectSize += threadCorrectSize;

#ifndef CLOSEOPENOMP 
#pragma omp critical 
#endif
            batchObjLoss += threadObjLoss;
        
        }  // end multi-processor

        NNet<XPU>::UpdateCumulateGrads(batchCumulatedGrads, &netsParas);
    }

    ShutdownTensorEngine<XPU>();
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;


    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            1)); // TODO ?

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }
}

State* GreedyChunker::decode(Instance *inst, NNetPara<XPU> &paras, State *lattice) {
    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transitionSystem.get());
    FeatureExtractor &featExtractor = *(m_featExtractor.get());
    FeatureEmbedding &fEmb = *(m_fEmb.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &paras));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

    InitTensorEngine<XPU>();
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        TensorContainer<cpu, 2, real_t> input;
        input.Resize(Shape2(1, num_in));

        TensorContainer<cpu, 2, real_t> pred;
        pred.Resize(Shape2(1, num_out));
       
        std::vector<std::vector<int>> featureVectors;
        featureVectors.resize(1);
        generateInputBatch(currentState, inst, featureVectors);
        fEmb.returnInput(featureVectors, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
        
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
    }

    ShutdownTensorEngine<XPU>();

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<std::vector<int>> &featvecs) {
    for (int i = 0; i < featvecs.size(); i++) {
        featvecs[i].resize(CConfig::nFeatureNum);
        m_featExtractor->extractFeature(*(state + i), *inst, featvecs[i]);
    }
}
