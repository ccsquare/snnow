#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

#ifdef DEBUG
 #define DEBUG1
#endif

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() {

}

double GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {

    auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
#ifdef DEBUGX
    std::cout << "longest instance's size: " << longestInst.size() << std::endl;
#endif
    State *lattice = new State[longestInst.size() + 1];

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(ChunkedSentence(currentInstance.input));

        State* predState = decode(&currentInstance, netsParas, lattice);

#ifdef DEBUGX
        std::cout << "Current instance's size: " << devInstances[inst].size() << std::endl;
        State *ptr = predState;
        int i = 1;
        while (ptr != nullptr && ptr->last_action != -1) {
            i++;
            ptr = ptr->previous_;
        }
        std::cout << "i = " << i << std::endl;
#endif 
        ChunkedSentence &predSent = predDevSet[inst];

#ifdef DEBUGX
        std::cout << "Before chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
        m_transitionSystem->generateOutput(*predState, predSent);

#ifdef DEBUGX
        std::cout << "After chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    delete []lattice;

#ifdef DEBUGX
    std::cout << "pred dev set's size: " << predDevSet.size() << std::endl;
    std::cout << "gold dev set's size: " << goldDevSet.size() << std::endl;
#endif
    auto res = Evalb::eval(predDevSet, goldDevSet);

    return std::get<2>(res);
}

void GreedyChunker::train(ChunkedDataSet &goldSet, InstanceSet &trainSet, InstanceSet &devSet) {
    initTrain(goldSet, trainSet);

    m_featExtractor->generateInstanceSetCache(devSet);
#ifdef DEBUGX
    std::cout << "After generateInstanceSetCache!" << std::endl;
#endif
    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);
#ifdef DEBUGX
    std::cout << "After readPretrainEmbeddings!" << std::endl;
#endif

    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(1, num_in, num_hidden, num_out);
    double bestDevFB1 = std::numeric_limits<double>::min();
#ifdef DEBGU1
    std::cout << "Before chunk training..." << std::endl;
#endif

    for (int iter = 0; iter < CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double currentFB1 = chunk(devSet, goldSet, netsParas);
            if (currentFB1 > bestDevFB1) {
                bestDevFB1 = currentFB1;
            }
            std::cout << "current iteration FB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentFB1 << "\t best FB1-score: " << bestDevFB1 << std::endl;
        }

        // random shuffle the training instances in the container,
        // and assign them for each thread
        std::vector<std::vector<GlobalExample *>> multiThread_miniBatch_data;

        // prepare mini-batch data for each threads
        std::random_shuffle(gExamples.begin(), gExamples.end());
        int threadExampleNum = std::min(CConfig::nBatchSize, static_cast<int>(gExamples.size())) / CConfig::nThread;
        auto sp = gExamples.begin();
        auto ep = sp + threadExampleNum;
        for (int i = 0; i < CConfig::nThread; i++) {
            std::vector<GlobalExample *> threadExamples;
            for (auto p = sp; p != ep; p++) {
                threadExamples.push_back(&(*p));
            }
#ifdef DEBUG3
            std::cout << "threadExamples' size: " << threadExamples.size() << std::endl;
            std::cout << "globalExamples' size: " << gExamples.size() << std::endl;
#endif
            sp = ep;
            ep += threadExampleNum;
            multiThread_miniBatch_data.push_back(threadExamples);
        }

#pragma omp parallel
        {
            std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));
            auto currentThreadData = multiThread_miniBatch_data[omp_get_thread_num()];
            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);

            auto longestSent = *std::max_element(currentThreadData.begin(), currentThreadData.end(), [](GlobalExample *g1, GlobalExample *g2) { return g1->instance.size() < g2->instance.size();} );
            int currentBatchSize = 0;
            for (GlobalExample* ge : currentThreadData) {
                currentBatchSize += ge->instance.size();
            }

            State *lattice = new State[longestSent->instance.size() + 1];

            // for evary instance in this mini-batch
            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                // fetch a to-be-trained instance
                GlobalExample *ge = currentThreadData[inst];
                int nMaxRound = ge->instance.size();

                lattice[0].clear();
                InitTensorEngine<XPU>();
                for (int nRound = 1; nRound <= nMaxRound; nRound++){
                    State *currentState = lattice + nRound - 1;
                    State *target = lattice + nRound;
                    int goldAct = ge->goldActs[nRound - 1];

                    TensorContainer<cpu, 2, real_t> input;
                    input.Resize(Shape2(1, num_in));

                    TensorContainer<cpu, 2, real_t> pred;
                    pred.Resize(Shape2(1, num_out));

                    std::vector<std::vector<int>> featureVectors(1);
                    generateInputBatch(currentState, &(ge->instance), featureVectors);
                    m_fEmb->returnInput(featureVectors, input);

                    nnet->Forward(input, pred, false);

                    std::vector<int> validActs;
                    m_transitionSystem->generateValidActs(*currentState, validActs);
                    int optAct = -1;
                    for (int i = 0; i < validActs.size(); i++) {
                        if (i == goldAct || validActs[i] >= 0) {
                            if (optAct == -1 || pred[0][i] > pred[0][optAct]){
                                optAct = i;
                            }
                        }
                    }
                    real_t maxScore = pred[0][optAct];
                    real_t goldScore = pred[0][goldAct];
                    real_t sum = 0.0;
                    for (int i = 0; i < validActs.size(); i++) {
                        if (i == goldAct || validActs[i] >= 0) {
                            pred[0][i] = exp(pred[0][i] - maxScore);
                            sum += pred[0][i];
                        }
                    }
                    for (int i = 0; i < validActs.size(); i++) {
                        if (i == goldAct || validActs[i] >= 0) {
                            pred[0][i] = pred[0][i] / sum;
                        } else {
                            pred[0][i] = 0.0;
                        }
                    }
                    pred[0][goldAct] -= 1.0;
                    for (int i = 0; i < validActs.size(); i++) {
                        pred[0][i] /= currentBatchSize;
                    }

                    nnet->Backprop(pred);
                    nnet->SubsideGrads(cumulatedGrads);

                    CScoredTransition trans;
                    trans(currentState, goldAct, currentState->score + goldScore);
                    *target = *currentState;
                    m_transitionSystem->move(*currentState, *target, trans);
                }
                ShutdownTensorEngine<XPU>();
            } // end for instance traverse

            delete []lattice;
#pragma omp barrier
#pragma omp critical
            NNet<XPU>::UpdateCumulateGrads(cumulatedGrads, &netsParas);
        }  // end multi-processor
    }
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;

    cout << "Training init..." << endl;
    cout << "  Training Instance num: " << trainSet.size() << endl;

    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

#ifdef DEBUG
    m_transitionSystem->displayLabel2ActionIdx();
#endif

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            1)); // TODO ?

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

#ifdef DEBUG1
    std::cout << "train set size: " << trainSet.size() << std::endl;
    std::cout << "dev gold set size: " << goldSet.size() << std::endl;
    std::cout << "global examples size: " << gExamples.size() << std::endl;
#endif
}

State* GreedyChunker::decode(Instance *inst, NNetPara<XPU> &paras, State *lattice) {
    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transitionSystem.get());
    FeatureExtractor &featExtractor = *(m_featExtractor.get());
    FeatureEmbedding &fEmb = *(m_fEmb.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &paras));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

    InitTensorEngine<XPU>();
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        TensorContainer<cpu, 2, real_t> input;
        input.Resize(Shape2(1, num_in));

        TensorContainer<cpu, 2, real_t> pred;
        pred.Resize(Shape2(1, num_out));
       
        std::vector<std::vector<int>> featureVectors;
        featureVectors.resize(1);
        generateInputBatch(currentState, inst, featureVectors);
        fEmb.returnInput(featureVectors, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = std::numeric_limits<real_t>::min();
        unsigned maxActID = std::numeric_limits<unsigned>::max();
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
#ifdef DEBUGX
        std::cout << "nRound = " << nRound << std::endl;
#endif
    }

    ShutdownTensorEngine<XPU>();

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<std::vector<int>> &featvecs) {
        for (int i = 0; i < featvecs.size(); i++) {
            featvecs[i].resize(CConfig::nFeatureNum);
            m_featExtractor->extractFeature(*(state + i), *inst, featvecs[i]);
        }
    }
