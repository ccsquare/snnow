#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() { } 

double GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {
    auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
    State *lattice = new State[longestInst.size() + 1];

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(ChunkedSentence(currentInstance.input));

        State* predState = decode(&currentInstance, netsParas, lattice);

        ChunkedSentence &predSent = predDevSet[inst];

        m_transitionSystem->generateOutput(*predState, predSent);
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cerr << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    delete []lattice;

    auto res = Evalb::eval(predDevSet, goldDevSet);

    return std::get<2>(res);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, NNetPara<XPU> &netsPara, double batchObjLoss, double posClassificationRate, double &bestDevFB1) {
    double currentFB1 = chunk(devSet, devGoldSet, netsPara);
    if (currentFB1 > bestDevFB1) {
        bestDevFB1 = currentFB1;
    }

    double loss = batchObjLoss;

    auto sf = std::cerr.flags();
    auto sp = std::cerr.precision();
    std::cerr.flags(std::ios::fixed);
    std::cerr.precision(2);
    std::cerr << "current iteration FB1-score: " << currentFB1 << "\tbest FB1-score: " << bestDevFB1 << std::endl;
    std::cerr << "current objective fun-score: " << loss << "\tclassfication rate: " << posClassificationRate << std::endl;
    std::cerr.flags(sf);
    std::cerr.precision(sp);
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    int exampleNumOfThread = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void display1Tensor( Tensor<cpu, 1, real_t> & tensor ){
    for(int i = 0; i < tensor.size(0); i++)
        std::cerr<<tensor[i]<<" ";
    std::cerr<<std::endl;
}

void display2Tensor( Tensor<cpu, 2, double> tensor ){
    std::cerr<<"size 0 :" << tensor.size(0)<<" size 1: "<<tensor.size(1)<<std::endl;
    for(int i = 0; i < tensor.size(0); i++){
       for(int j = 0; j < tensor.size(1); j++)
           std::cerr<<tensor[i][j]<<" ";
       std::cerr<<std::endl;
    }
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cerr << "Initing FeatureManager & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cerr << "Excuting devset generateInstanceSetCache & readPretrainEmbeddings..." << std::endl;
    m_featManager->generateInstanceSetCache(devSet);
    std::cerr << "  Greedy train set size: " << trainExamplePtrs.size() << std::endl;
    m_featManager->readPretrainEmbeddings(CConfig::strEmbeddingPath);

    const static int num_in = m_featManager->totalFeatSize;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;
    const static int batchSize = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size()));

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    InitTensorEngine<XPU>();

    NNetPara<XPU> netsParas(1, num_in, num_hidden, num_out);

    double bestDevFB1 = -1.0;

    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;

    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            printEvaluationInfor(devSet, devGoldSet, netsParas, batchObjLoss, posClassificationRate, bestDevFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;

        // prepare mini-batch data for each threads
        std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);
        UpdateGrads<XPU> batchCumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);


#pragma omp parallel
        {
            int threadIndex = omp_get_thread_num();
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];

            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;

            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
            std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));

            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {

                Example *e = currentThreadData[inst];

                TensorContainer<cpu, 2, real_t> input;
                input.Resize(Shape2(1, num_in));

                TensorContainer<cpu, 2, real_t> pred;
                pred.Resize(Shape2(1, num_out));

                std::vector<FeatureVector> featureVectors;
                featureVectors.push_back(e->features);
                m_featManager->returnInput(featureVectors, input, 1);

                nnet->Forward(input, pred, false);

                std::vector<int> validActs(e->labels);

                int optAct = -1;
                int goldAct = -1;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        if (optAct == -1 || pred[0][i] > pred[0][optAct]){
                            optAct = i;
                        }

                        if (validActs[i] == 1) {
                            goldAct = i;
                        }
                    }
                }
                if (optAct == goldAct) {
                    threadCorrectSize += 1;
                }

                real_t maxScore = pred[0][optAct];
                real_t goldScore = pred[0][goldAct];

                real_t sum = 0.0;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = std::exp(pred[0][i] - maxScore);
                        sum += pred[0][i];
                    }
                }

                threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;

                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = pred[0][i] / sum;
                    } else {
                        pred[0][i] = 0.0;
                    }
                }
                pred[0][goldAct] -= 1.0;

                for (int i = 0; i < validActs.size(); i++) {
                    pred[0][i] /= batchSize;
                }

                nnet->Backprop(pred);
                nnet->SubsideGrads(cumulatedGrads);
            }

#pragma omp barrier
#pragma omp critical 
            {
                batchCumulatedGrads.cg_hbias = batchCumulatedGrads.cg_hbias + cumulatedGrads.cg_hbias;
                batchCumulatedGrads.cg_Wi2h = batchCumulatedGrads.cg_Wi2h + cumulatedGrads.cg_Wi2h;
                batchCumulatedGrads.cg_Wh2o = batchCumulatedGrads.cg_Wh2o + cumulatedGrads.cg_Wh2o;
            }

#pragma omp critical 
            batchCorrectSize += threadCorrectSize;

#pragma omp critical 
            batchObjLoss += threadObjLoss;
        
        }  // end multi-processor

        NNet<XPU>::UpdateCumulateGrads(batchCumulatedGrads, &netsParas);
    }

    ShutdownTensorEngine<XPU>();
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cerr;
    using std::endl;

    m_featManager.reset(new FeatureManager());
    m_featManager->init(goldSet, CConfig::fInitRange);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featManager->getKnownLabels());

    m_featManager->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }
}

State* GreedyChunker::decode(Instance *inst, NNetPara<XPU> &paras, State *lattice) {
    const static int num_in = m_featManager->totalFeatSize;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    FeatureManager &fManager = *(m_featManager.get());
    ActionStandardSystem &tranSystem = *(m_transitionSystem.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &paras));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

    InitTensorEngine<XPU>();
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        TensorContainer<cpu, 2, real_t> input;
        input.Resize(Shape2(1, num_in));

        TensorContainer<cpu, 2, real_t> pred;
        pred.Resize(Shape2(1, num_out));
       
        std::vector<FeatureVector> featureVectors;
        featureVectors.push_back(FeatureVector(fManager.featTypes, fManager.featEmbs));
        generateInputBatch(currentState, inst, featureVectors);
        fManager.returnInput(featureVectors, input, 1);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
        
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
    }

    ShutdownTensorEngine<XPU>();

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<FeatureVector> &featvecs) {
    for (int i = 0; i < featvecs.size(); i++) {
        m_featManager->extractFeature(*(state + i), *inst, featvecs[i]);
    }
}
