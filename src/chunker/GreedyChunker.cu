#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

#ifdef DEBUG
#define DEBUG1
// #define DEBUG2
// #define DEBUG3
// #define DEBUG4
// #define DEBUG5
// #define DEBUG6
// #define DEBUG7
// #define CONSTROUNDDEBUG
// #define ADDREGURLOSS
#define DEBUG8
// #define DEBUG9
#endif

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() {

}

double GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {

    auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
#ifdef DEBUGX
    std::cout << "longest instance's size: " << longestInst.size() << std::endl;
#endif
    State *lattice = new State[longestInst.size() + 1];

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(ChunkedSentence(currentInstance.input));

        State* predState = decode(&currentInstance, netsParas, lattice);

        ChunkedSentence &predSent = predDevSet[inst];

#ifdef DEBUG9
        std::cout << "Before chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
        m_transitionSystem->generateOutput(*predState, predSent);

#ifdef DEBUG9
        std::cout << "After chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    delete []lattice;

#ifdef DEBUGX
    std::cout << "pred dev set's size: " << predDevSet.size() << std::endl;
    std::cout << "gold dev set's size: " << goldDevSet.size() << std::endl;
#endif
    auto res = Evalb::eval(predDevSet, goldDevSet);

    return std::get<2>(res);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, NNetPara<XPU> &netsPara, double batchObjLoss, double posClassificationRate, double &bestDevFB1) {
    static int iter = 0;
    iter++;
#ifdef DEBUG9
    std::cout << "iter = " << iter << std::endl;
#endif
    double currentFB1 = chunk(devSet, devGoldSet, netsPara);
    if (currentFB1 > bestDevFB1) {
        bestDevFB1 = currentFB1;
    }

    double loss = batchObjLoss;
#ifdef ADDREGURLOSS
    double paraLoss = 0.0;
    for (int ii = 0; ii < netsPara.Wi2h.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wi2h.shape_[1]; jj++) {
            paraLoss += netsPara.Wi2h[ii][jj] * netsPara.Wi2h[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.Wh2o.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wh2o.shape_[1]; jj++) {
            paraLoss += netsPara.Wh2o[ii][jj] * netsPara.Wh2o[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.hbias.shape_[0]; ii++) {
        paraLoss += netsPara.hbias[ii] * netsPara.hbias[ii];
    }
    paraLoss *= 0.5 * CConfig::fRegularizationRate;

    loss += paraLoss;
#endif

    std::cout << "current iteration FB1-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << currentFB1 << "\tbest FB1-score: " << bestDevFB1 << std::endl;
    std::cout << "current objective fun-score: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << loss << "\tclassfication rate: " << std::setiosflags(std::ios::fixed) << std::setprecision(2) << posClassificationRate << std::endl;
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    int exampleNumOfThread = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cout << "Initing FeatureExtractor & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cout << "Excuting generateInstanceSetCache & readPretrainEmbeddings..." << std::endl;
    m_featExtractor->generateInstanceSetCache(devSet);

    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;
    const static int batchSize = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size()));

    // omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(1, num_in, num_hidden, num_out);

    double bestDevFB1 = -1.0;

    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;
    for (int iter = 1; iter <= CConfig::nRound; iter++) {
        std::cout << "\nPress any key:" << std::endl;
        char tch;
        std::cin >> tch;
        if (iter % CConfig::nEvaluatePerIters == 0) {
            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            printEvaluationInfor(devSet, devGoldSet, netsParas, batchObjLoss, posClassificationRate, bestDevFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;
        // std::cout << "initialize multiThread_miniBatch_data" << std::endl;

        // prepare mini-batch data for each threads
        // std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);

// #pragma omp parallel
        {
            // auto currentThreadData = multiThread_miniBatch_data[omp_get_thread_num()];
            auto currentThreadData = multiThread_miniBatch_data[0];
            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;

            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
                Example *e = currentThreadData[inst];
                std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));

                InitTensorEngine<XPU>();
                TensorContainer<cpu, 2, real_t> input;
                input.Resize(Shape2(1, num_in));

                TensorContainer<cpu, 2, real_t> pred;
                pred.Resize(Shape2(1, num_out));

                std::vector<std::vector<int>> featureVectors;
                featureVectors.push_back(e->features);
                m_fEmb->returnInput(featureVectors, input);
#ifdef DEBUG8
                std::cout << "Feature input index: ";
                for (int i = 0; i < CConfig::nFeatureNum; i++) {
                    std::cout << e->features[i] << " ";
                }
                std::cout << std::endl;
#endif

                nnet->Forward(input, pred, false);

                std::vector<int> validActs(e->labels);
#ifdef DEBUG8
                std::cout << "[valid acts sequence]: ";
                for (int ai = 0; ai < validActs.size(); ai++){
                    std::cout << validActs[ai] << " ";
                }
                std::cout << std::endl;
                // std::cout <<"[nn scores]: ";
                // for (int ai = 0; ai < validActs.size(); ai++) {
                //     std::cout << pred[0][ai] << " ";
                // }
                // std::cout << std::endl;
#endif 
                int optAct = -1;
                int goldAct = -1;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        if (optAct == -1 || pred[0][i] > pred[0][optAct]){
                            optAct = i;
                        }

                        if (validActs[i] == 1) {
                            goldAct = i;
                        }
                    }
                }
                if (optAct == goldAct) {
                    threadCorrectSize += 1;
                }

                real_t maxScore = pred[0][optAct];
                real_t goldScore = pred[0][goldAct];
#ifdef DEBUG8
                std::cout << "maxAct = " << optAct << "\tgoldAct = " << goldAct << std::endl;
                std::cout << "maxScore = " << maxScore << "\tgoldScore = " << goldScore << std::endl;
#endif
                real_t sum = 0.0;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = std::exp(pred[0][i] - maxScore);
                        sum += pred[0][i];
                    }
                }

#ifdef DEBUG8
                // std::cout << "sum = " << sum << "\tlog(sum) = " << std::log(sum) << "\tlog(gold) = " << std::log(std::exp(goldScore - maxScore)) << std::endl;
#endif

                threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;
#ifdef DEBUG8
                // std::cout <<"[divided by exp(maxScore)]: ";
                // for (int ai = 0; ai < validActs.size(); ai++) {
                //     std::cout << pred[0][ai] << " ";
                // }
                // std::cout << std::endl;
                // std::cout << "threadObjLoss: " << threadObjLoss << std::endl;
#endif
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = pred[0][i] / sum;
                    } else {
                        pred[0][i] = 0.0;
                    }
                }
                pred[0][goldAct] -= 1.0;
#ifdef DEBUG8
                std::cout <<"[probability]: ";
                for (int ai = 0; ai < validActs.size(); ai++) {
                    std::cout << pred[0][ai] << " ";
                }
                std::cout << std::endl;
#endif
                nnet->Backprop(pred);
                nnet->SubsideGrads(cumulatedGrads);

                ShutdownTensorEngine<XPU>();
            }

// #pragma omp barrier
// #pragma omp critical 
            {
                NNet<XPU>::UpdateCumulateGrads(cumulatedGrads, &netsParas);

                batchCorrectSize += threadCorrectSize;

                batchObjLoss += threadObjLoss;

            }
        }  // end multi-processor

//             auto longestSent = *std::max_element(currentThreadData.begin(), currentThreadData.end(), [](GlobalExample *g1, GlobalExample *g2) { return g1->instance.size() < g2->instance.size();} );
// 
//             State *lattice = new State[longestSent->instance.size() + 1];
// 
//             // for every instance in this mini-batch
//             for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {
//                 // fetch a to-be-trained instance
//                 GlobalExample *ge = currentThreadData[inst];
//                 int nMaxRound = static_cast<int>(ge->instance.size());
// 
//                 lattice[0].clear();
//                 for (int i = 0; i <= nMaxRound; i++) {
//                     lattice[i].m_nLen = nMaxRound;
//                 }
// 
//                 InitTensorEngine<XPU>();
//                 for (int nRound = 1; nRound <= nMaxRound; nRound++){
//                     std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));
// #ifdef CONSTROUNDDEBUG
//                     const int roundConstant = 16;
//                     if (nRound != roundConstant) 
//                         continue;
// #endif
//                     State *currentState = lattice + nRound - 1;
//                     State *target = lattice + nRound;
//                     int goldAct = ge->goldActs[nRound - 1];
// 
//                     TensorContainer<cpu, 2, real_t> input;
//                     input.Resize(Shape2(1, num_in));
// 
//                     TensorContainer<cpu, 2, real_t> pred;
//                     pred.Resize(Shape2(1, num_out));
// 
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant)
// #endif
//                     std::cout << "iter = " << iter << ", inst = " << inst << ", nRound = " << nRound << std::endl; 
// #endif
//                     std::vector<std::vector<int>> featureVectors(1);
//                     generateInputBatch(currentState, &(ge->instance), featureVectors);
//                     m_fEmb->returnInput(featureVectors, input);
// 
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant){
// #endif
//                     std::cout << "Feature input value: ";
//                     for (int i = 0; i < CConfig::nFeatureNum; i++) {
//                         std::cout << input[0][i] << " ";
//                     }
//                     std::cout << std::endl;
// #ifdef CONSTROUNDDEBUG
//                     }
// #endif
// #endif
//                     nnet->Forward(input, pred, false);
// 
//                     std::vector<int> validActs;
//                     m_transitionSystem->generateValidActs(*currentState, validActs);
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant){
// #endif
//                     std::cout << "[valid acts sequence]: ";
//                     for (int ai = 0; ai < validActs.size(); ai++){
//                         std::cout << validActs[ai] << " ";
//                     }
//                     std::cout << std::endl;
//                     std::cout <<"[nn scores]: ";
//                     for (int ai = 0; ai < validActs.size(); ai++) {
//                         std::cout << pred[0][ai] << " ";
//                     }
//                     std::cout << std::endl;
// #ifdef CONSTROUNDDEBUG
//                     }
// #endif
// #endif
// 
//                     int optAct = -1;
//                     for (int i = 0; i < validActs.size(); i++) {
//                         if (i == goldAct || validActs[i] >= 0) {
//                             if (optAct == -1 || pred[0][i] > pred[0][optAct]){
//                                 optAct = i;
//                             }
//                         }
//                     }
//                     if (optAct == goldAct) {
//                         threadCorrectSize += 1;
//                     }
// 
//                     real_t maxScore = pred[0][optAct];
//                     real_t goldScore = pred[0][goldAct];
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant) {
// #endif
//                     std::cout << "maxAct = " << optAct << "\tgoldAct = " << goldAct << std::endl;
//                     std::cout << "maxScore = " << maxScore << "\tgoldScore = " << goldScore << std::endl;
// #ifdef CONSTROUNDDEBUG
//                     }
// #endif
// #endif
//                     real_t sum = 0.0;
//                     for (int i = 0; i < validActs.size(); i++) {
//                         if (i == goldAct || validActs[i] >= 0) {
//                             pred[0][i] = std::exp(pred[0][i] - maxScore);
//                             sum += pred[0][i];
//                         }
//                     }
// #ifdef DEBUG2
//                     std::cout << "log(sum) = " << std::log(sum) << "\tlog(gold) = " << std::log(std::exp(goldScore - maxScore)) << std::endl;
// #endif
// 
//                     threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;
// 
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant) {
// #endif
//                     std::cout << "sum = " << sum << std::endl;
//                     std::cout <<"[divided by exp(maxScore)]: ";
//                     for (int ai = 0; ai < validActs.size(); ai++) {
//                         std::cout << pred[0][ai] << " ";
//                     }
//                     std::cout << std::endl;
//                     std::cout << "threadObjLoss: " << threadObjLoss << std::endl;
// #ifdef CONSTROUNDDEBUG
//                     }
// #endif
// #endif
// 
//                     for (int i = 0; i < validActs.size(); i++) {
//                         if (i == goldAct || validActs[i] >= 0) {
//                             pred[0][i] = pred[0][i] / sum;
//                         } else {
//                             pred[0][i] = 0.0;
//                         }
//                     }
//                     pred[0][goldAct] -= 1.0;
// 
// #ifdef DEBUG7
// #ifdef CONSTROUNDDEBUG
//                     if (nRound == roundConstant) {
// #endif
//                     std::cout <<"[probability]: ";
//                     for (int ai = 0; ai < validActs.size(); ai++) {
//                         std::cout << pred[0][ai] << " ";
//                     }
//                     std::cout << std::endl;
// #ifdef CONSTROUNDDEBUG
//                     }
// #endif
// #endif
// 
//                     for (int i = 0; i < validActs.size(); i++) {
//                         pred[0][i] /= batchSize;
//                     }
// 
//                     nnet->Backprop(pred);
//                     nnet->SubsideGrads(cumulatedGrads);
// 
//                     CScoredTransition trans(currentState, goldAct, currentState->score + goldScore);
//                     *target = *currentState;
//                     m_transitionSystem->move(*currentState, *target, trans);
//                 }
//                 ShutdownTensorEngine<XPU>();
//             } // end for instance traverse
// 
//             delete []lattice;
// 
// #pragma omp barrier
// #pragma omp critical 
//             {
//                 NNet<XPU>::UpdateCumulateGrads(cumulatedGrads, &netsParas);
// 
//                 batchCorrectSize += threadCorrectSize;
// 
//                 batchObjLoss += threadObjLoss;
// 
//             }
//         }  // end multi-processor
    }
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;


    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

#ifdef DEBUGX
    m_transitionSystem->displayLabel2ActionIdx();
#endif

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            1)); // TODO ?

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

#ifdef DEBUGX
    std::cout << "train set size: " << trainSet.size() << std::endl;
    std::cout << "dev gold set size: " << goldSet.size() << std::endl;
    std::cout << "global examples size: " << gExamples.size() << std::endl;
#endif
    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }
}

State* GreedyChunker::decode(Instance *inst, NNetPara<XPU> &paras, State *lattice) {
    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transitionSystem.get());
    FeatureExtractor &featExtractor = *(m_featExtractor.get());
    FeatureEmbedding &fEmb = *(m_fEmb.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &paras));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

#ifdef DEBUG5
    std::cout << "[deco action sequances]: ";
#endif
    InitTensorEngine<XPU>();
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        TensorContainer<cpu, 2, real_t> input;
        input.Resize(Shape2(1, num_in));

        TensorContainer<cpu, 2, real_t> pred;
        pred.Resize(Shape2(1, num_out));
       
        std::vector<std::vector<int>> featureVectors;
        featureVectors.resize(1);
        generateInputBatch(currentState, inst, featureVectors);
        fEmb.returnInput(featureVectors, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
#ifdef DEBUG6
        for (unsigned ii = 0; ii < validActs.size(); ++ii) {
            std::cout << "validActs[" << ii << "]=" << validActs[ii] << " "; 
        }
        std::cout << std::endl;
        for (unsigned ii = 0; ii < pred.shape_[1]; ii++) {
            std::cout << "pred[" << ii << "]=" << pred[0][ii] << " ";
        }
        std::cout << std::endl;
#endif
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

#ifdef DEBUG5
        maxActID = example.goldActs[nRound - 1];
        std::cout << maxActID << " ";
#endif
        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
#ifdef DEBUG5
        std::cout << "[" << target->last_action << "](" << trans.action << ") ";
#endif
#ifdef DEBUGX
        std::cout << "nRound = " << nRound << std::endl;
#endif
    }

#ifdef DEBUG5
    std::cout << std::endl;
#endif

    ShutdownTensorEngine<XPU>();

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<std::vector<int>> &featvecs) {
        for (int i = 0; i < featvecs.size(); i++) {
            featvecs[i].resize(CConfig::nFeatureNum);
            m_featExtractor->extractFeature(*(state + i), *inst, featvecs[i]);
        }
    }
