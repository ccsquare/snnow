#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: GreedyChunker.cpp
	> Author: cheng chuan
	> Mail: cc.square0@gmail.com 
	> Created Time: Mon 07 Dec 2015 08:56:14 PM CST
 ************************************************************************/
#include <ctime>
#include <omp.h>
#include <random>
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <limits.h>

#include "Config.h"

#include "Evalb.h"

#include "GreedyChunker.h"

#define DEBUG

#ifdef DEBUG
#define DEBUG1
// #define DEBUG2
// #define DEBUG3
// #define DEBUG4
// #define DEBUG5
// #define DEBUG6
// #define DEBUG7
// #define CONSTROUNDDEBUG
// #define ADDREGURLOSS
// #define DEBUG8
// #define DEBUG9
// #define DEBUG10
// #define DEBUG11
// #define DEBUG12
#endif

GreedyChunker::GreedyChunker() {

}

GreedyChunker::GreedyChunker(bool isTrain) {
    m_bTrain = isTrain;
}

GreedyChunker::~GreedyChunker() {

}

double GreedyChunker::chunk(InstanceSet &devInstances, ChunkedDataSet &goldDevSet, NNetPara<XPU> &netsParas) {

    auto longestInst = *std::max_element(devInstances.begin(), devInstances.end(), [](Instance &inst1, Instance &inst2) { return inst1.size() < inst2.size();} );
#ifdef DEBUGX
    std::cout << "longest instance's size: " << longestInst.size() << std::endl;
#endif
    State *lattice = new State[longestInst.size() + 1];

    clock_t start, end;
    start = clock();
    ChunkedDataSet predDevSet;
    for (unsigned inst = 0; inst < devInstances.size(); inst++) {
        Instance &currentInstance = devInstances[inst];
        predDevSet.push_back(ChunkedSentence(currentInstance.input));

        State* predState = decode(&currentInstance, netsParas, lattice);

        ChunkedSentence &predSent = predDevSet[inst];

#ifdef DEBUG9
        std::cout << "Before chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
        m_transitionSystem->generateOutput(*predState, predSent);

#ifdef DEBUG10
        std::cout << "After chunked: " << std::endl;
        std::cout << predSent << std::endl;
#endif
    }
    end = clock();

    double time_used = (double)(end - start) / CLOCKS_PER_SEC;
    std::cout << "totally chunk " << devInstances.size() << " sentences, time: " << time_used << " average: " << devInstances.size() / time_used << " sentences/second!" << std::endl;

    delete []lattice;

#ifdef DEBUGX
    std::cout << "pred dev set's size: " << predDevSet.size() << std::endl;
    std::cout << "gold dev set's size: " << goldDevSet.size() << std::endl;
#endif
    auto res = Evalb::eval(predDevSet, goldDevSet);

    return std::get<2>(res);
}

void GreedyChunker::printEvaluationInfor(InstanceSet &devSet, ChunkedDataSet &devGoldSet, NNetPara<XPU> &netsPara, double batchObjLoss, double posClassificationRate, double &bestDevFB1) {
    double currentFB1 = chunk(devSet, devGoldSet, netsPara);
    if (currentFB1 > bestDevFB1) {
        bestDevFB1 = currentFB1;
    }

    double loss = batchObjLoss;
#ifdef ADDREGURLOSS
    double paraLoss = 0.0;
    for (int ii = 0; ii < netsPara.Wi2h.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wi2h.shape_[1]; jj++) {
            paraLoss += netsPara.Wi2h[ii][jj] * netsPara.Wi2h[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.Wh2o.shape_[0]; ii++) {
        for (int jj = 0; jj < netsPara.Wh2o.shape_[1]; jj++) {
            paraLoss += netsPara.Wh2o[ii][jj] * netsPara.Wh2o[ii][jj];
        }
    }
    for (int ii = 0; ii < netsPara.hbias.shape_[0]; ii++) {
        paraLoss += netsPara.hbias[ii] * netsPara.hbias[ii];
    }
    std::cout << "current |W|^2: " << paraLoss << std::endl;
    paraLoss *= 0.5 * CConfig::fRegularizationRate;

    loss += paraLoss;
#endif

    auto sf = std::cout.flags();
    auto sp = std::cout.precision();
    std::cout.flags(std::ios::fixed);
    std::cout.precision(2);
    std::cout << "current iteration FB1-score: " << currentFB1 << "\tbest FB1-score: " << bestDevFB1 << std::endl;
    std::cout << "current objective fun-score: " << loss << "\tclassfication rate: " << posClassificationRate << std::endl;
    std::cout.flags(sf);
    std::cout.precision(sp);
}

void GreedyChunker::generateMultiThreadsMiniBatchData(std::vector<ExamplePtrs> &multiThread_miniBatch_data) {
    int exampleNumOfThread = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size())) / CConfig::nThread;

    auto sp = trainExamplePtrs.begin();
    auto ep = sp + exampleNumOfThread;
    for (int i = 0; i < CConfig::nThread; i++) {
        ExamplePtrs threadExamples;

        for (auto p = sp; p != ep; p++) {
            threadExamples.push_back(*p);
        }

        multiThread_miniBatch_data.push_back(threadExamples);

        sp = ep;
        ep += exampleNumOfThread;
    }
}

void GreedyChunker::train(ChunkedDataSet &trainGoldSet, InstanceSet &trainSet, ChunkedDataSet &devGoldSet, InstanceSet &devSet) {
    std::cout << "Initing FeatureExtractor & ActionStandardSystem & generateTrainingExamples..." << std::endl;
    initTrain(trainGoldSet, trainSet);

    std::cout << "Excuting generateInstanceSetCache & readPretrainEmbeddings..." << std::endl;
    m_featExtractor->generateInstanceSetCache(devSet);

    m_featExtractor->readPretrainEmbeddings(CConfig::strEmbeddingPath, *m_fEmb);

    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;
    const static int batchSize = std::min(CConfig::nBatchSize, static_cast<int>(trainExamplePtrs.size()));

    omp_set_num_threads(CConfig::nThread);

    srand(0);

    NNetPara<XPU> netsParas(1, num_in, num_hidden, num_out);

    double bestDevFB1 = -1.0;

    int batchCorrectSize = 0;
    double batchObjLoss = 0.0;

    InitTensorEngine<XPU>();

    for (int iter = 1; iter <= CConfig::nRound; iter++) {
#ifdef DEBUG13
        std::cout << "iter = " << iter << std::endl;
        char tch;
        std::cin >> tch;
#endif
        if (iter % CConfig::nEvaluatePerIters == 0) {

            double posClassificationRate = 100 * static_cast<double>(batchCorrectSize) / batchSize;

            printEvaluationInfor(devSet, devGoldSet, netsParas, batchObjLoss, posClassificationRate, bestDevFB1);
        }
        batchCorrectSize = 0;
        batchObjLoss = 0.0;

        // random shuffle the training instances in the container,
        // and assign them for each threads
        std::vector<ExamplePtrs> multiThread_miniBatch_data;
        // std::cout << "initialize multiThread_miniBatch_data" << std::endl;

        // prepare mini-batch data for each threads
        // std::random_shuffle(trainExamplePtrs.begin(), trainExamplePtrs.end());
        generateMultiThreadsMiniBatchData(multiThread_miniBatch_data);
        UpdateGrads<XPU> batchCumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
#ifdef DEBUG13
        for (int ii = 0; ii < batchCumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
            for (int jj = 0; jj < batchCumulatedGrads.cg_Wi2h.shape_[1]; jj++) {
                batchCumulatedGrads.cg_Wi2h[ii][jj] = 0.0;
            }
        }
        for (int ii = 0; ii < batchCumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
            for (int jj = 0; jj < batchCumulatedGrads.cg_Wh2o.shape_[1]; jj++) {
                batchCumulatedGrads.cg_Wh2o[ii][jj] = 0.0;
            }
        }
        for (int ii = 0; ii < batchCumulatedGrads.cg_hbias.shape_[0]; ii++) {
            batchCumulatedGrads.cg_hbias[ii] = 0.0;
        }
#endif
#ifdef DEBUG8
            int threadconst = 0;
#endif
        
// #pragma omp parallel
        {
            int threadIndex = omp_get_thread_num();
            auto currentThreadData = multiThread_miniBatch_data[threadIndex];
            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);
#ifdef DEBUG13
            for (int ii = 0; ii < cumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
                for (int jj = 0; jj < cumulatedGrads.cg_Wi2h.shape_[1]; jj++) {
                    cumulatedGrads.cg_Wi2h[ii][jj] = 0.0;
                }
            }
            for (int ii = 0; ii < cumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
                for (int jj = 0; jj < cumulatedGrads.cg_Wh2o.shape_[1]; jj++) {
                    cumulatedGrads.cg_Wh2o[ii][jj] = 0.0;
                }
            }
            for (int ii = 0; ii < cumulatedGrads.cg_hbias.shape_[0]; ii++) {
                cumulatedGrads.cg_hbias[ii] = 0.0;
            }
#endif

            int threadCorrectSize = 0;
            double threadObjLoss = 0.0;


            for (unsigned inst = 0; inst < currentThreadData.size(); inst++) {

                Example *e = currentThreadData[inst];
                std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &netsParas));

                TensorContainer<cpu, 2, real_t> input;
                input.Resize(Shape2(1, num_in));

                TensorContainer<cpu, 2, real_t> pred;
                pred.Resize(Shape2(1, num_out));

                std::vector<std::vector<int>> featureVectors;
                featureVectors.push_back(e->features);
                m_fEmb->returnInput(featureVectors, input);
#ifdef DEBUG8
                if (threadIndex == threadconst) {
                    std::cout << "Feature input index: ";
                    for (int i = 0; i < CConfig::nFeatureNum; i++) {
                        std::cout << e->features[i] << " ";
                    }
                    std::cout << std::endl;
                }
#endif

                nnet->Forward(input, pred, false);

                std::vector<int> validActs(e->labels);
#ifdef DEBUG8
                if (threadIndex == threadconst) {
                    std::cout << "[valid acts sequence]: ";
                    for (int ai = 0; ai < validActs.size(); ai++){
                        std::cout << validActs[ai] << " ";
                    }
                    std::cout << std::endl;
                    std::cout <<"[nn scores]: ";
                    for (int ai = 0; ai < validActs.size(); ai++) {
                        std::cout << pred[0][ai] << " ";
                    }
                    std::cout << std::endl;
                }
#endif 
                int optAct = -1;
                int goldAct = -1;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        if (optAct == -1 || pred[0][i] > pred[0][optAct]){
                            optAct = i;
                        }

                        if (validActs[i] == 1) {
                            goldAct = i;
                        }
                    }
                }
                if (optAct == goldAct) {
                    threadCorrectSize += 1;
                }

                real_t maxScore = pred[0][optAct];
                real_t goldScore = pred[0][goldAct];
#ifdef DEBUG8
                if (threadIndex == threadconst) {
                    std::cout << "maxAct = " << optAct << "\tgoldAct = " << goldAct << std::endl;
                    std::cout << "maxScore = " << maxScore << "\tgoldScore = " << goldScore << std::endl;
                }
#endif
                real_t sum = 0.0;
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = std::exp(pred[0][i] - maxScore);
                        sum += pred[0][i];
                    }
                }

                threadObjLoss += (std::log(sum) - (goldScore - maxScore)) / batchSize;
#ifdef DEBUG8
                // std::cout <<"[divided by exp(maxScore)]: ";
                // for (int ai = 0; ai < validActs.size(); ai++) {
                //     std::cout << pred[0][ai] << " ";
                // }
                // std::cout << std::endl;
                // std::cout << "threadObjLoss: " << threadObjLoss << std::endl;
#endif
                for (int i = 0; i < validActs.size(); i++) {
                    if (validActs[i] >= 0) {
                        pred[0][i] = pred[0][i] / sum;
                    } else {
                        pred[0][i] = 0.0;
                    }
                }
                pred[0][goldAct] -= 1.0;
#ifdef DEBUG8
                if (threadIndex == threadconst) {
                    std::cout <<"[probability]: ";
                    for (int ai = 0; ai < validActs.size(); ai++) {
                        std::cout << pred[0][ai] << " ";
                    }
                    std::cout << std::endl;
                }
#endif

                for (int i = 0; i < validActs.size(); i++) {
                    pred[0][i] /= batchSize;
                }

                nnet->Backprop(pred);
                nnet->SubsideGrads(cumulatedGrads);
#ifdef DEBUG13
                std::cout << "inst = " << inst << std::endl;
                std::cout << "check cumulatedGrads..." << std::endl;
                bool nanAppear = false;
                for (int ii = 0; ii < cumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wi2h.shape_[1]; jj++) {
                        if (isnan(cumulatedGrads.cg_Wi2h[ii][jj])) {
                            std::cout << "NaN appears in cumulatedGrads.cg_Wi2h!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wh2o.shape_[1]; jj++) {
                        if (isnan(cumulatedGrads.cg_Wh2o[ii][jj])) {
                            std::cout << "NaN appears in cumulatedGrads.cg_Wh2o!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_hbias.shape_[0]; ii++) {
                    if (isnan(cumulatedGrads.cg_hbias[ii])) {
                        std::cout << "NaN appears in cumulatedGrads.cg_hbias!" << std::endl;
                        nanAppear = true;
                    }
                }
                if (!nanAppear)
                    std::cout << "No NaN appears in cumulatedGrads!" << std::endl;
#endif
            }

// #pragma omp barrier
// #pragma omp critical 
            {
#ifdef DEBUG13
                std::cout << "check batch cumulatedGrads..." << std::endl;
                bool nanAppear = false;
                for (int ii = 0; ii < cumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wi2h.shape_[1]; jj++) {
                        if (isnan(cumulatedGrads.cg_Wi2h[ii][jj])) {
                            std::cout << "NaN appears in cumulatedGrads.cg_Wi2h!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
                    for (int jj = 0; jj < cumulatedGrads.cg_Wh2o.shape_[1]; jj++) {
                        if (isnan(cumulatedGrads.cg_Wh2o[ii][jj])) {
                            std::cout << "NaN appears in cumulatedGrads.cg_Wh2o!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < cumulatedGrads.cg_hbias.shape_[0]; ii++) {
                    std::cout << cumulatedGrads.cg_hbias[ii] << " ";
                    if (isnan(cumulatedGrads.cg_hbias[ii])) {
                        std::cout << "NaN appears in cumulatedGrads.cg_hbias!" << std::endl;
                        nanAppear = true;
                    }
                }
                std::cout << std::endl;
                if (!nanAppear)
                    std::cout << "No NaN appears in cumulatedGrads!" << std::endl;
                std::cout << "Before adding cumulatedGrads.cg_hbias" << std::endl;
                for (int ii = 0; ii < batchCumulatedGrads.cg_hbias.shape_[0]; ii++) {
                    std::cout << batchCumulatedGrads.cg_hbias[ii] << " ";
                    if (isnan(batchCumulatedGrads.cg_hbias[ii])) {
                        std::cout << "NaN appears in batchCumulatedGrads.cg_hbias!" << std::endl;
                        nanAppear = true;
                    }
                }
                std::cout << std::endl;
#endif

                batchCumulatedGrads.cg_hbias += cumulatedGrads.cg_hbias;
                batchCumulatedGrads.cg_Wi2h += cumulatedGrads.cg_Wi2h;
                batchCumulatedGrads.cg_Wh2o += cumulatedGrads.cg_Wh2o;
#ifdef DEBUG13
                std::cout << "check batchCumulatedGrads..." << std::endl;
                nanAppear = false;
                for (int ii = 0; ii < batchCumulatedGrads.cg_Wi2h.shape_[0]; ii++) {
                    for (int jj = 0; jj < batchCumulatedGrads.cg_Wi2h.shape_[1]; jj++) {
                        if (isnan(batchCumulatedGrads.cg_Wi2h[ii][jj])) {
                            std::cout << "NaN appears in batchCumulatedGrads.cg_Wi2h!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < batchCumulatedGrads.cg_Wh2o.shape_[0]; ii++) {
                    for (int jj = 0; jj < batchCumulatedGrads.cg_Wh2o.shape_[1]; jj++) {
                        if (isnan(batchCumulatedGrads.cg_Wh2o[ii][jj])) {
                            std::cout << "NaN appears in batchCumulatedGrads.cg_Wh2o!" << std::endl;
                            nanAppear = true;
                        }
                    }
                }
                for (int ii = 0; ii < batchCumulatedGrads.cg_hbias.shape_[0]; ii++) {
                    std::cout << batchCumulatedGrads.cg_hbias[ii] << " ";
                    if (isnan(batchCumulatedGrads.cg_hbias[ii])) {
                        std::cout << "NaN appears in batchCumulatedGrads.cg_hbias!" << std::endl;
                        nanAppear = true;
                    }
                }
                std::cout << std::endl;
                if (!nanAppear)
                    std::cout << "No NaN appears in batchCumulatedGrads!" << std::endl;
#endif
            }

// #pragma omp critical 
            batchCorrectSize += threadCorrectSize;

// #pragma omp critical 
            batchObjLoss += threadObjLoss;
        }  // end multi-processor

        NNet<XPU>::UpdateCumulateGrads(batchCumulatedGrads, &netsParas);
    }

    ShutdownTensorEngine<XPU>();
}

void GreedyChunker::initTrain(ChunkedDataSet &goldSet, InstanceSet &trainSet) {
    using std::cout;
    using std::endl;


    m_featExtractor.reset(new FeatureExtractor());
    m_featExtractor->getDictionaries(goldSet);

    m_transitionSystem.reset(new ActionStandardSystem());
    m_transitionSystem->makeTransition(m_featExtractor->getKnownLabels());

#ifdef DEBUGX
    m_transitionSystem->displayLabel2ActionIdx();
#endif

    m_fEmb.reset(new FeatureEmbedding(m_featExtractor->size(),
            CConfig::nFeatureNum,
            CConfig::nEmbeddingDim,
            1)); // TODO ?

    m_featExtractor->generateTrainingExamples(*(m_transitionSystem.get()), trainSet, goldSet, gExamples);

#ifdef DEBUGX
    std::cout << "train set size: " << trainSet.size() << std::endl;
    std::cout << "dev gold set size: " << goldSet.size() << std::endl;
    std::cout << "global examples size: " << gExamples.size() << std::endl;
#endif
    for (auto &gExample : gExamples) {
        for (auto &example : gExample.examples) {
            trainExamplePtrs.push_back(&(example));
        }
    }
}

State* GreedyChunker::decode(Instance *inst, NNetPara<XPU> &paras, State *lattice) {
    const static int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const static int num_hidden = CConfig::nHiddenSize;
    const static int num_out = m_transitionSystem->nActNum;

    int nSentLen = inst->input.size();
    int nMaxRound = nSentLen;
    ActionStandardSystem &tranSystem = *(m_transitionSystem.get());
    FeatureExtractor &featExtractor = *(m_featExtractor.get());
    FeatureEmbedding &fEmb = *(m_fEmb.get());
    std::shared_ptr<NNet<XPU>> nnet(new NNet<XPU>(1, num_in, num_hidden, num_out, &paras));

    State *retval = nullptr;
    for (int i = 0; i < nMaxRound + 1; ++i) {
        lattice[i].m_nLen = nSentLen;
    }

    lattice[0].clear();

#ifdef DEBUG5
    std::cout << "[deco action sequances]: ";
#endif
    // InitTensorEngine<XPU>();
    for (int nRound = 1; nRound <= nMaxRound; nRound++){
        State *currentState = lattice + nRound - 1;
        State *target = lattice + nRound;

        TensorContainer<cpu, 2, real_t> input;
        input.Resize(Shape2(1, num_in));

        TensorContainer<cpu, 2, real_t> pred;
        pred.Resize(Shape2(1, num_out));
       
        std::vector<std::vector<int>> featureVectors;
        featureVectors.resize(1);
        generateInputBatch(currentState, inst, featureVectors);
        fEmb.returnInput(featureVectors, input);

        nnet->Forward(input, pred, false);
        
        std::vector<int> validActs;
        tranSystem.generateValidActs(*currentState, validActs);
        // get max-score valid action
        real_t maxScore = 0.0;
        unsigned maxActID = 0;
#ifdef DEBUG6
        for (unsigned ii = 0; ii < validActs.size(); ++ii) {
            std::cout << "validActs[" << ii << "]=" << validActs[ii] << " "; 
        }
        std::cout << std::endl;
        for (unsigned ii = 0; ii < pred.shape_[1]; ii++) {
            std::cout << "pred[" << ii << "]=" << pred[0][ii] << " ";
        }
        std::cout << std::endl;
#endif
        for (unsigned actID = 0; actID < validActs.size(); ++actID) {
            if (validActs[actID] == -1) {
                continue;
            }

            if (actID == 0 || pred[0][actID] > maxScore) {
                maxScore = pred[0][actID];
                maxActID = actID;
            }
        }

#ifdef DEBUG5
        maxActID = example.goldActs[nRound - 1];
        std::cout << maxActID << " ";
#endif
        CScoredTransition trans(currentState, maxActID, currentState->score + maxScore);
        *target = *currentState;
        tranSystem.move(*currentState, *target, trans);
        retval = target;
#ifdef DEBUG5
        std::cout << "[" << target->last_action << "](" << trans.action << ") ";
#endif
#ifdef DEBUGX
        std::cout << "nRound = " << nRound << std::endl;
#endif
    }

#ifdef DEBUG5
    std::cout << std::endl;
#endif

    // ShutdownTensorEngine<XPU>();

    return retval;
}

void GreedyChunker::generateInputBatch(State *state, Instance *inst, std::vector<std::vector<int>> &featvecs) {
        for (int i = 0; i < featvecs.size(); i++) {
            featvecs[i].resize(CConfig::nFeatureNum);
            m_featExtractor->extractFeature(*(state + i), *inst, featvecs[i]);
        }
    }
