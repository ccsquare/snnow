#include "hip/hip_runtime.h"
//
// Created by zhouh on 16-4-5.
//
#include <ctime>
#include <memory>

#include "DepParser.h"

using namespace mshadow;
using namespace mshadow::expr;

/**
 * initialize the feature extractor and
 * transition system handlers for parser
 */
DepParser::DepParser(bool bTrain){
    beamSize = FLAGS_beam_size;
    be_train = bTrain;
    trainsition_system_ptr.reset(new DepArcStandardSystem());
    feature_extractor_ptr.reset(new DepParseFeatureExtractor());
}

/**
 *  do the training init for the formal training
 *
 *  0. init the feature type of this system
 *  1. get dictionary for the feature extractor
 *  2. init the transition system handler for the parser
 *  3. init the feature embedding handler
 */
void DepParser::trainInit(DataSet &training_data) {

    std::clog << "======================================";
    std::clog << "Training Init!" << std::endl;
    std::clog << "Training Instance Num: " << training_data.getSize() << std::endl;
    std::clog << "======================================";


    std::clog << "###Begin to init the feature types of this system: " << std::endl;


    // prepare the handler for parsing
    std::clog << "###Begin to init the dictionaries!" << std::endl;
    feature_extractor_ptr->getDictionaries(training_data);  // dictionary for feature index
    feature_extractor_ptr->displayDict();
    std::clog << "###End to init the dictionaries!" << std::endl;

    std::clog << "###Begin to create feature types!" << std::endl;
    FeatureTypes feature_types;
    FeatureType word_feat_type(FeatureType::c_word_type_name,
                               DepParseFeatureExtractor::feature_nums[DepParseFeatureExtractor::c_word_dict_index],
                               feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_word_dict_index]->size(),
                               c_word_feature_dim);
    FeatureType tag_feat_type(FeatureType::c_tag_type_name,
                              DepParseFeatureExtractor::feature_nums[DepParseFeatureExtractor::c_tag_dict_index],
                              feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_tag_dict_index]->size(),
                              c_tag_feature_dim);
    FeatureType label_feat_type(FeatureType::c_tag_type_name,
                                DepParseFeatureExtractor::feature_nums[DepParseFeatureExtractor::c_dep_label_dict_index],
                                feature_extractor_ptr->dictionary_ptrs_table[DepParseFeatureExtractor::c_dep_label_dict_index]->size(),
                                c_label_feature_dim);
    feature_types.push_back(word_feat_type);
    feature_types.push_back(tag_feat_type);
    feature_types.push_back(label_feat_type);

    // set the feature types for feature handlers
    DepParseFeatureExtractor::setFeatureTypes(feature_types);
    FeatureVector::setFeatureTypes(feature_types);
    std::clog << "###End to create feature types!" << std::endl;


    // init transition system
    std::clog << "###Init the transition system!" << std::endl;
    trainsition_system_ptr->makeTransition(feature_extractor_ptr->getKnownDepLabelVector(),
                                           feature_extractor_ptr->getKnownDepLabelVectorMap());

    std::clog << "###Begin to generate the training examples!" << std::endl;
    feature_extractor_ptr->generateGreedyTrainingExamples(trainsition_system_ptr, training_data, greedy_example_ptrs);
    std::clog << "Constructing dictionary and training examples done!" << std::endl;
}

void DepParser::train(DataSet &train_data, DataSet &dev_data) {

    /*
     * prepare for the neural networks, every parsing step maintains a specific net
     * because each parsing step has different updating gradients.
     */
    const int num_in = feature_extractor_ptr->getTotalInputSize();
    const int num_hidden = FLAGS_hidden_size;
    const int num_out = DepParseShiftReduceActionFactory::total_action_num;
    const int beam_size = FLAGS_beam_size;
    const int batch_size = std::min(FLAGS_batch_size, static_cast<int>(greedy_example_ptrs.size()));
    const bool be_dropout = FLAGS_dropout_prob;

    trainInit(train_data);

    /*
     * create the model for training
     */
    std::clog << "###Begin to construct training model." << std::endl;
    Model<cpu> model(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, NULL);
    Model<cpu> adagrad_squares(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types,
                               NULL);  // for adagrad updating
    Stream <gpu> *stream = stream = NewStream<gpu>();
    std::clog << "###End to construct training model." << std::endl;


    double best_uas = -1;
    for (int iter = 1; iter <= FLAGS_max_training_iteration_num; iter++) {



        // record the cost time
        auto start = std::chrono::high_resolution_clock::now();

        // random shuffle the training instances in the container,
        // get the shuffled training data for the mini-batch training of this iteration
        std::random_shuffle(greedy_example_ptrs);
        int batch_example_index_end = std::min(batch_size, static_cast<int>(trainExamplePtrs.size()));
        std::vector<std::shared_ptr<Example>> multiThread_miniBatch_data(greedy_example_ptrs.begin(),
                                                                         batch_example_index_end);

        // cumulated gradients for updating
        Model<cpu> batch_cumulated_grads(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, NULL);

        // copy from the parameter model to current model
        Copy(modelPtr->Wi2h, paraModel.Wi2h, stream);
        Copy(modelPtr->Wh2o, paraModel.Wh2o, stream);
        Copy(modelPtr->hbias, paraModel.hbias, stream);

        Model<gpu> gradients(num_in, num_hidden, num_out, feature_extractor_ptr->feature_types, stream);

        // create the neural net for prediction
        std::shared_ptr<FeedForwardNNet<gpu>> nnet;
        nnet.rest(new FeedForwardNNet<gpu>(batch_size, num_in, num_hidden, num_out, model));

        FeatureVectors feature_vectors(multiThread_miniBatch_data.size());

        TensorContainer<cpu, 2, real_t> input(Shape2(batch_size, num_in));

        std::vector<std::vector<int>> valid_action_vectors(multiThread_miniBatch_data.size());

        TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(batch_size, num_out));

        /*
         * init the input and predict output
         */
        input = 0.0;
        batch_predict_output = 0.0;

        // fill the feature vectors for batch training


        // prepare batch training data!
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {
            auto e = greedy_example_ptrs[inst];

            feature_vectors[inst] = e->feature_vector;
            valid_action_vectors[insti] = e->predict_label;
        }

        feature_extractor_ptr->returnInput(featureVectors, model.featEmbs, input);

        nnet->Forward(input, batch_predict_output, CConfig::bDropOut);


        int total_correct_predict_action_num = 0;
        double loss = 0;
        for (int inst = 0; inst < multiThread_miniBatch_data.size(); inst++) {

            int opt_act = -1;
            int gold_act = -1;

            std::vector<int> &valid_acts = valid_action_vectors[inst];

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }

                    if (valid_acts[i] == 1) {
                        gold_act = i;
                    }
                }
            }

            if (opt_act == gold_act) {
                total_correct_predict_action_num += 1;
            }

            real_t max_score = batch_predict_output[inst][opt_act];
            real_t gold_score = pred[insti][gold_act];

            real_t sum = 0.0;

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    batch_predict_output[inst][i] = std::exp(batch_predict_output[inst][i] - max_score);
                    sum += batch_predict_output[inst][i];
                }
            }

            loss += (std::log(sum) - (gold_score - max_score)) / multiThread_miniBatch_data.size();

            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    batch_predict_output[insti][i] = batch_predict_output[insti][i] / sum;
                } else {
                    batch_predict_output[insti][i] = 0.0;
                }
            }
            batch_predict_output[insti][goldAct] -= 1.0;
        }

        batch_predict_output /= static_cast<real_t>(multiThread_miniBatch_data.size());

        nnet->Backprop(batch_predict_output);
        nnet->SubsideGradsTo(&gradients, feature_vectors);

        model.update(&batch_cumulated_grads, &adagrad_squares);
        auto end = std::chrono::high_resolution_clock::now();

        if (iter % FLAGS_evaluate_per_iteration == 0) {
            double time_used = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / MICROSECOND;
            std::clog << "[" << iter << "] totally train " << batch_size << " examples, time: " << time_used <<
            " average: " << batch_size / time_used << " examples/second!" << std::endl;
        }

        /*
         * do the evaluation in iteration of training
         * save the best resulting model
         */
        if (iter % FLAGS_evaluate_per_iteration == 0) {
            // do the evaluation
            double dev_uas = test(dev_data, model, nnet);
            if (dev_uas > best_uas)
                saveModel(FLAGS_model_file);
        }
    }


}

//===============================================================================



double DepParser::test(DataSet &test_data, Model<gpu> & model, FeedForwardNNet<gpu> & net) {

    auto trees = test_data.outputs;
    trees = static_cast<std::vector<std::shared_ptr<DepParseTree>> >(trees);
    auto inputs = test_data.inputs;
    inputs = static_cast<std::vector<std::shared_ptr<DepParseInput>> >(inputs);

    const int num_in = feature_extractor_ptr->getTotalInputSize();
    const int num_hidden = FLAGS_hidden_size;
    const int num_out = DepParseShiftReduceActionFactory::total_action_num;

    std::vector<DepTree> predict_trees(test_data.size);

    for (int inst = 0; inst < test_data.size; ++inst) {

        auto &input_ptr_i = inputs[inst];
        auto &tree_ptr_i = trees[inst];

        // n shift and n reduce, one more reduce action for root
        int total_act_num_one_sentence = (input_ptr_i->size() - 1) * 2;

        /*
         * cache the dependency label in the training set
         */
        std::vector<int> labelIndexCache(tree_ptr_i.size);
        int index = 0;
        for (auto iter = tree_ptr_i.nodes.begin(); iter != tree_ptr_i.nodes.end();
             iter++) {
            int labelIndex = getLabelIndex(iter->label);

            if (labelIndex == -1) {
                std::cerr << "Dep label " << iter->label
                << " is not in labelMap!" << std::endl;
                exit(1);
            }

            labelIndexCache[index] = labelIndex;
            index++;
        }

        std::shared_ptr<DepParseState> state_ptr;
        state_ptr.reset(new DepParseState());

        state_ptr->len_ = input_ptr_i->size();
        state_ptr->initCache();
        getCache(input_ptr_i.operator*());

        //for every state of a sentence
        for (int j = 0; !state_ptr->complete(); j++) {

            TensorContainer<cpu, 2, real_t> input(Shape2(1, num_in));
            TensorContainer<cpu, 2, real_t> batch_predict_output(Shape2(1, num_out));

            /*
             * init the input and predict output
             */
            input = 0.0;
            batch_predict_output = 0.0;

            std::vector<int> valid_acts(total_act_num_one_sentence, 0);

            //get current state features
            std::shared_ptr<FeatureVector> fv = feature_extractor_ptr->getFeatureVectors(*state_ptr, *input_ptr_i);
            FeatureVectors fvs;
            fvs.push_back(*fv);

            //get current state valid actions
            trainsition_system_ptr->getValidActs(*state, valid_acts);

            feature_extractor_ptr->returnInput(fvs, model.featEmbs, input);

            net->Forward(input, batch_predict_output, CConfig::bDropOut);


            int opt_Act = -1;
            for (int i = 0; i < valid_acts.size(); i++) {
                if (valid_acts[i] >= 0) {
                    if (opt_act == -1 || batch_predict_output[inst][i] > batch_predict_output[inst][opt_act]) {
                        opt_act = i;
                    }
                }
            }

            transit_system_ptr->Move(*state_ptr, DepParseShiftReduceActionFactory::action_table[opt_act]);
        }


        // generate the predict tree from the complete state
        trainsition_system_ptr->GenerateOutput( *state_ptr, input_ptr_i, predict_trees[inst] );
    }

    auto result = DepParseEvalb::evalb(predict_trees, trees);
    return result.first * 100;
}