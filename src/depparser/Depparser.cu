#include "hip/hip_runtime.h"
/*
 * Depparser.cpp
 *
 *  Created on: Jul 2, 2015
 *      Author: zhouh
 */
#include <ctime>
#include <memory>
#include "Depparser.h"

using namespace mshadow;
using namespace mshadow::expr;

Depparser::Depparser(bool bTrain) {
    beamSize = CConfig::nBeamSize;
    m_bTrain = bTrain;
    transitionSystem = new ArcStandardSystem();
}

Depparser::~Depparser() {
    delete fEmb;
    delete transitionSystem;
}

void Depparser::trainInit(std::vector<Instance> & trainInstances, std::vector<DepTree> & goldTrees){

    std::cout<<"Training begin!"<<std::endl;
    std::cout<<"Training Instance Num: "<<trainInstances.size()<<std::endl;

    /*Prepare the feature extractor*/
    featExtractor.getDictionaries(goldTrees);
    transitionSystem->makeTransition(featExtractor.knowLabels);
    featExtractor.displayDict();
    fEmb = new FeatureEmbedding(featExtractor.getDicSize(), CConfig::nFeatureNum, CConfig::nEmbeddingDim, beamSize);
    featExtractor.generateTrainingExamples(transitionSystem, trainInstances, goldTrees, gExamples);
    /*
     * prepare the feature embedding and fill in pre-train embedding
     */
    featExtractor.readPretrainEmbeddings( CConfig::strEmbeddingPath, *fEmb );

    std::cout << "Constructing dictionary and training examples done!"<<std::endl;
}

void Depparser::train(std::vector<Instance> & trainInstances, std::vector<DepTree> & goldTrees,
        std::vector<Instance> & devInstances, std::vector<DepTree> & devTrees) {
    
    trainInit(trainInstances, goldTrees);
    featExtractor.getInstancesCache(devInstances); // train instances get cache in get training examples
    /*
     * prepare for the neural networks, every parsing step maintains a specific net
     * because each parsing step has different updating gradients.
     */
    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = transitionSystem->nActNum;
    const int beamSize = CConfig::nBeamSize;
    omp_set_num_threads(CConfig::nThread);  //set the threads for mini-batch learning
    srand(0);
    // std::shared_ptr< NNetPara<XPU> > ptrNetsParas(new NNetPara<XPU>(beamSize, num_in, num_hidden, num_out));
    NNetPara<XPU> netsParas(beamSize, num_in, num_hidden, num_out);
    double bestdevUAS = -1.0;

    // for every iteration
    for(int iter = 0; iter < CConfig::nRound; iter++){

        /*
         * Evaluate per iterations
         */
        if( (iter % CConfig::nEvaluatePerIters) == 0 ){
            double currentUAS = parse( devInstances, devTrees, netsParas );
            if( currentUAS > bestdevUAS )
                bestdevUAS = currentUAS;
            std::cout<<"current iteration UAS: "<<currentUAS<<" new best UAS:\t"<< bestdevUAS<<std::endl;;
        }

        /*
         * randomly sample the training instances in the container,
         * and assign them for each thread
         */
        std::vector<std::vector<GlobalExample*>> multiThread_miniBtach_data;

        //get mini-batch data for each threads
        std::random_shuffle ( gExamples.begin(), gExamples.end() );
        int threadExampleNum = CConfig::nBatchSize / CConfig::nThread;
        auto sp = gExamples.begin();
        auto ep = sp + threadExampleNum;
        for(int i = 0; i < CConfig::nThread; i++){
            std::vector<GlobalExample*> threadExamples;
            for(auto p = sp; p != ep; p++)
                threadExamples.push_back( &( *p ) );
            sp = ep;
            ep += threadExampleNum;
            multiThread_miniBtach_data.push_back(threadExamples);
        }

        /*std::cout<<"begin to create cuda!"<<std::endl;*/
        
        // begin to multi-thread training
#pragma omp parallel
        {
            /*hipSetDevice( omp_get_thread_num() % 4 );*/
            auto currentThreadData = multiThread_miniBtach_data[omp_get_thread_num()];
            UpdateGrads<XPU> cumulatedGrads(netsParas.stream, num_in, num_hidden, num_out);

            //for every instance
            for(unsigned inst = 0; inst < currentThreadData.size(); inst++){
                //get current training instance
                GlobalExample * example =  currentThreadData[inst];

                TensorContainer<gpu,2, real_t> mask;
                mask.set_stream(netsParas.stream);
                netsParas.rnd.SampleUniform(&mask, 0.0f, 1.0f);

                TNNets tnnets( beamSize, num_in, num_hidden, num_out, &netsParas);
                /*
                 * decoding and updating
                 */
                std::cout<<"begin to decod!"<<std::endl;
                BeamDecodor decodor( &( example->instance ), beamSize, true );
                
                std::cout<<"end to decod!"<<std::endl;
                State * predState = decodor.decoding( transitionSystem, tnnets, featExtractor, *fEmb, example );
                tnnets.updateTNNetParas( cumulatedGrads, decodor.beam, decodor.bEarlyUpdated, decodor.nGoldTransitIndex, decodor.goldScoredTran );
                
                /*std::cout<<"begin to back subsidegrads!"<<std::endl;*/
            } // instance #for end

            /*std::cout<<"Begin to update cumulated grads!"<<std::endl;*/
#pragma omp barrier
#pragma omp critical
            NNet<XPU>::UpdateCumulateGrads(cumulatedGrads, &netsParas);

        } // end multi-processor
    } // iteration #for end
}

double Depparser::parse( std::vector<Instance> & devInstances, std::vector<DepTree> & devTree, NNetPara<XPU> & netsParas){

            const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
            const int num_hidden = CConfig::nHiddenSize;
            const int num_out = transitionSystem->nActNum;
            const int beamSize = CConfig::nBeamSize;
            TNNets tnnets( beamSize, num_in, num_hidden, num_out, &netsParas, false );

            std::vector<DepTree> predTrees(devInstances.size());

            clock_t start, end; 
            start = clock();
            //for every instance
            for(unsigned inst = 0; inst < devInstances.size(); inst++){
                predTrees[inst].init(devInstances[inst].input);
                /*std::cout<<"instance\t"<<inst<<std::endl;*/
                //get current training instance
                BeamDecodor decodor( &( devInstances[inst] ), beamSize, false );
                State * predState = decodor.decoding( transitionSystem, tnnets, featExtractor, *fEmb);
                transitionSystem->GenerateOutput( *predState, devInstances[inst].input, predTrees[inst] );
            } // instance #for end
            end = clock(); 
            double timeuse = (double)(end - start) / CLOCKS_PER_SEC;
            std::cout<<"totally parse "<<devInstances.size()<<" sentences, time : "<< timeuse << " average: "<< devInstances.size()/timeuse<<" sentences/second!"<<std::endl;
            /*
             * evaluate
             */
            auto result = Evalb::evalb(predTrees, devTree);

            return result.first * 100;
}
