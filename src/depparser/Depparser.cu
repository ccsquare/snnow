#include "hip/hip_runtime.h"
/*
 * Depparser.cpp
 *
 *  Created on: Jul 2, 2015
 *      Author: zhouh
 */

#include <omp.h>
#include <random>
#include <algorithm>

#include "Depparser.h"
#include "State.h"
#include "Config.h"
#include "mshadow/tensor.h"
#include "NNet.h"
#include "FeatureEmbedding.h"
#include "FeatureExtractor.h"

using namespace mshadow;
using namespace mshadow::expr;

Depparser::Depparser(bool bTrain) {
	/*beamSize = CConfig::nBeamSize;*/
	/*m_bTrain = bTrain;*/
}

Depparser::~Depparser() {
}

void Depparser::train(std::vector<DepParseInput> inputs, std::vector<DepTree> goldTrees,
        std::vector<DepParseInput> devInputs, std::vector<DepTree> devTrees) {

    std::cout<<"Training begin!"<<std::endl;
    std::cout<<"Training Instance Num: "<<inputs.size()<<std::endl;

    /*Prepare the feature extractor*/
    featExtractor.getDictionaries(goldTrees);
    featExtractor.generateTrainingExamples(inputs, goldTrees, gExamples);

    // prepare for the neural networks, every parsing step maintains a specific net
    // because each parsing step has different updating gradients.
    const int num_in = CConfig::nEmbeddingDim * CConfig::nFeatureNum;
    const int num_hidden = CConfig::nHiddenSize;
    const int num_out = kActNum;
    const int beamSize = CConfig::nBeamSize;
    omp_set_num_threads(CConfig::nThread);  //set the threads for mini-batch learning
    srand(0);
    NNet<gpu>::init(beamSize, num_in, num_hidden, num_out);	//init the static member in the neural net
    FeatureEmbedding<cpu> fEmb(CConfig::nFeatureNum, CConfig::nEmbeddingDim, beamSize);

    // for every iteration
    for(int iter = 0; iter < CConfig::nRound; iter++){
        /*
         * randomly sample the training instances in the container,
         * and assign them for each thread
         */
        std::vector<std::vector<GlobalExample*>> multiThread_miniBtach_data;

        //get mini-batch data for each threads
        std::random_shuffle ( gExamples.begin(), gExamples.end() );
        int threadExampleNum = CConfig::nBatchSize / CConfig::nThread;
        auto sp = gExamples.begin();
        auto ep = sp + threadExampleNum;
        for(int i = 0; i < CConfig::nThread; i++){
            std::vector<GlobalExample*> threadExamples;
            for(auto p = sp; p != ep; p++){
                threadExamples.push_back( &( *p ) );
            }
            sp = ep;
            ep += threadExampleNum;
            multiThread_miniBtach_data.push_back(threadExamples);
        }

        //set up mshadow tensor
        InitTensorEngine<gpu>();

        // begin to multi-thread training
#pragma omp parallel
        {
            auto currentThreadData = multiThread_miniBtach_data[omp_get_thread_num()];

            // temp input layer
            TensorContainer<cpu, 2> input;
            input.Resize( Shape2( beamSize, num_in ) );
            // temp output layer
            TensorContainer<cpu, 2> pred;
            pred.Resize( Shape2( beamSize, num_out ) );

            //for every instance
            for(unsigned inst = 0; inst < currentThreadData.size(); inst++){
                //get current training instance
                GlobalExample * example =  currentThreadData[inst];
                const int sentLen = example->wordIdx.size();
                const int maxRound = sentLen * 2 + 1;
                const int max_lattice_size =  (beamSize + 1) * maxRound;
//				int num_results = 0;
                int round = 0;
                int currentBeamSize = 1; // initially, the beam only have one empty state
                int correctStateIdx;
                bool bBeamContainGold = true;
                double maxScore = 0;
                Beam beam(beamSize);

                std::vector<NNet<gpu>*> nets;

                if(inst % 1000 == 0)
                    std::cout<<"Processing sentence "<<inst<<std::endl;
                // beam search decoding
                State * lattice = new State[max_lattice_size];
                State * lattice_index[maxRound];
                State * correctState = lattice;
                for (int i = 0; i < max_lattice_size; ++i) {
                    lattice[i].len_ = sentLen;
                }

                lattice[0].clear();
                lattice[0].setBeamIdx(0);
                correctState = lattice;
                lattice_index[0] = lattice;
                lattice_index[1] = lattice_index[0] + 1;

                // for every round in training
//				int beamIdx = 0;
                for(round = 1; round < maxRound; round++){

                    NNet<gpu> *net = new NNet<gpu>(beamSize, num_in, num_hidden, num_out);
                    nets.push_back(net);
                    // new round, set beam gold false
                    bBeamContainGold = false;
                    // extract feature vectors in batch
                    std::vector<std::vector<int> > featureVectors(currentBeamSize);
                    getInputBatch(lattice_index[round - 1], example->wordIdx,
                                  example->tagIdx, featureVectors);
                    fEmb.returnInput(featureVectors, input);
                    net->Forward(input, pred);

                    // for every state in the last beam, expand and insert into next beam
                    int stateIdx = 0;
                    for (State * currentState = lattice_index[round - 1];
                            currentState != lattice_index[round]; ++currentState, ++stateIdx) {
                        std::vector<int> validActs;
                        currentState->getValidActs(validActs);

                        //for every valid action
                        for(unsigned actID = 0; actID < validActs.size(); ++actID){
                            //skip invalid action
                            if(validActs[actID] == -1)
                                continue;
                            //construct scored transition, and insert into beam
                            CScoredTransition trans;
                            trans(currentState, actID, currentState->score + pred[stateIdx][actID]);
                            beam.insert(trans);
                            currentBeamSize = ( currentBeamSize + 1 ) >= beamSize ? beamSize : ( currentBeamSize + 1 );
                        } // valid action #for end

                        //lazy expand the states in the beam
                        for (int i = 0; i < beam.currentBeamSize; ++i) {
                            const CScoredTransition& transition = beam.beam[i];
                            State* target = lattice_index[round] + i;
                            target->copy( *(transition.source) );
                            // generate candidate state according to the states in beam
                            target->Move(transition.action);
                            target->setBeamIdx(i);
                            target->score = transition.score;
                            target->previous_ = transition.source;
                            target->bGold = target->previous_->bGold & target->last_action == example->goldActs[round - 1]; // beam states contain gold state ?  bBeamContainGold |= target->bGold;

                            if(target->bGold == true){
                                correctState = target;
                                correctStateIdx = i;
                            }
                            if( i == 0 || target->score > maxScore )
                                maxScore = target->score;
                        }
                    } // beam #for end

                    if( bEarlyUpdate & !bBeamContainGold & m_bTrain)
                        break;

                    // prepare lattice for next parsing round
                    lattice_index[round + 1] = lattice_index[round] + currentBeamSize;
                } //round #for end

                // update parameter
                if (m_bTrain) {

                    std::vector<State*> trainingStates;
                    for(int bi = 0; bi < currentBeamSize; ++bi){
                        trainingStates.push_back( beam.beam[bi].source );
                    }
                    /* With early update, now the gold state fall out beam,*/
                    /* we need to expand the gold state one more step.*/
                    if( bEarlyUpdate & !bBeamContainGold ){
                        State* next_correct_state = lattice_index[round] + currentBeamSize;
                        next_correct_state->copy(*correctState);
                        next_correct_state->Move(example->goldActs[round - 1]);
                        next_correct_state->previous_ = correctState;
                        correctState = next_correct_state;
                        //endLatice = correctState;
                        correctStateIdx = currentBeamSize;
                        trainingStates.emplace_back(correctState);
                    }
                    /*computes the gradients of beam contrastive learning*/
                    int trainingDataSize = trainingStates.size();
                    std::vector<float> updateParas(trainingDataSize, 0); // updating parameter vector
                    // softmax
                    double sum =0;
                    for (int b_j = 0; b_j < trainingDataSize; b_j++) {
                        updateParas[b_j] = exp( trainingStates[b_j]->score - maxScore );
                        sum += updateParas[b_j];
                    }
                    for (int b_j = 0; b_j < trainingDataSize; b_j++) {
                        updateParas[b_j] = updateParas[b_j] / sum;
                        sum += updateParas[b_j];
                    }
                    updateParas[correctStateIdx] -= 1.0;

                    /*  Back propagation updating,*/
                    /*  from last parsing state to the former states*/
                    for(int backRound = round; backRound > 0; --backRound){
                        TensorContainer<cpu, 2> grads;
                        input.Resize(Shape2(beamSize, num_out));
                        int i = 0;
                        for(auto iter = trainingStates.begin(); iter != trainingStates.end(); iter++, i++){
                            grads[ ( *iter )->previous_->beamIdx ][ ( *iter )->last_action ] = updateParas[i];
                            *iter = ( *iter )->previous_;
                        }
                        nets[backRound - 1]->Backprop(grads);
                    }

                    NNet<gpu>::Update();

                } // updating end
                else{ // in testing
                    // get best expanded state
                    State * bestState = lattice_index[round];
                    for (State * p = lattice_index[round]; p != lattice_index[round + 1]; ++p) {
                        if (bestState->score < p->score) {
                            bestState = p;
                        }
                    }
                } // testing end

            } // instance #for end

        } // end multi-processor
        ShutdownTensorEngine<gpu>();

    } // iteration #for end

}

void Depparser::parse(std::vector<DepParseInput> inputs) {
}


   //get the feature vector in all the beam states,
   //and return the input layer of neural network in a batch.
void Depparser::getInputBatch(State* state, std::vector<int>& wordIndexCache,
        std::vector<int>& tagIndexCache,
        std::vector<std::vector<int> >& featvecs) {

    for(unsigned i = 0; i < featvecs.size(); i++){
        std::vector<int> featvec(CConfig::nFeatureNum);
        featExtractor.featureExtract( state + i, wordIndexCache, tagIndexCache, featvec);
        featvecs.push_back(featvec);
    }
}


